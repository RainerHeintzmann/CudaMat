#include "hip/hip_runtime.h"
/************************* CudaMat ******************************************
 *   Copyright (C) 2008-2009 by Rainer Heintzmann                          *
 *   heintzmann@gmail.com                                                  *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; Version 2 of the License.               *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************
 * Compile with:
 * Windows:
system('"c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin\vcvars32.bat"')
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin')

Window 64 bit:
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I"c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" ')

Linux:
 * File sudo vi /usr/local/cuda/bin/nvcc.profile
 * needs the flag -fPIC  in the include line
system('nvcc -c cudaArith.cu -v -I/usr/local/cuda/include/')
 */

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stddef.h>
#include <math.h>

#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#include "hipfft/hipfft.h"
#include "cudaArith.h"
#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

//#define BLOCKSIZE 512
//#define BLOCKSIZE 512
// below is blocksize for temporary array for reduce operations. Has to be a power of 2 in size
#ifndef CUIMAGE_REDUCE_THREADS  // this can be defined at compile time via the flag NVCCFLAG='-D CUIMAGE_REDUCE_THREADS=512'
#define CUIMAGE_REDUCE_THREADS 512
#endif
// (prop.maxThreadsPerBlock)
// #define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 128
//#define CUIMAGE_REDUCE_BLOCKS  64

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

#define NBLOCKSL(N,blockSize) 1
// min((N/blockSize+(N%blockSize==0?0:1)),prop.maxGridSize[0])


#define MemoryLayout(N,blockSize,nBlocks)	blockSize=prop.maxThreadsPerBlock; \
{ size_t numb=NBLOCKS(N,blockSize);                    \
    if (numb<prop.maxGridSize[0])                   \
    nBlocks.x=numb;                                 \
else                                                \
    {nBlocks.x=(size_t)(sqrt((float)numb)+1);          \
    nBlocks.y=(size_t)(sqrt((float)numb)+1);}}

// the real part is named ".x" and the imaginary ".y" in the hipfftComplex datatype
__device__ hipfftComplex cuda_resultVal;   // here real and complex valued results can be stored to be then transported to the host
__device__ size_t cuda_resultInt;   // here size-valued results can be stored to be then transported to the host
static ACCU_ARRTYPE * TmpRedArray=0;   // This temporary array will be constructed on the device, whenever the first reduce operation is performed
static ACCU_ARRTYPE * accum = 0;       // This is the corresponding array on the host side
static size_t CurrentRedSize=0;    // Keeps track of how much reduce memory is allocated on the device
static const int MinRedBlockSize=65536;    // defines the chunks of memory (in floats) which will be used in reduce operations
static struct hipDeviceProp_t prop;  // Defined in cudaArith.h: contains the cuda Device properties. is set during initialisation
    // prop.maxThreadsPerBlock;  // 512
    // prop.multiProcessorCount;   // 30
    // prop.warpSize;   // 32
    // prop.maxThreadsDim[0];   // 512  = max blocksize
    // prop.maxGridSize[0];   // 65535  = max GridSize = max nBlocks?


#define mysumpos(a,b) ((a)+((b)>0))
#define mysum(a,b) ((a)+(b))
#define maxCond(a,b) (((b)>(a)))
#define minCond(a,b) (((b)<(a)))

#define Sqr(a) ((a)*(a))

#define sign(x) (((x) > 0) - ((x) < 0))

// below are code snippets used in other macros 
#define Coords3DFromIdx(idx,sSize)                                      \
  size_t x=(idx)%sSize.s[0];                                               \
  size_t y=(idx/sSize.s[0])%sSize.s[1];                                    \
  size_t z=(idx/(sSize.s[0]*sSize.s[1]))%sSize.s[2];                       

#define IdxFromCoords3D(x,y,z,dSize,dOffs) \
  unsigned size_t idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]);  \


#define CoordsNDFromIdx(idx,sSize,pos)                                  \
   IntND pos;                                               \
   { size_t resid=idx;                                               \
  for(int _d=0;_d<CUDA_MAXDIM;_d++)                                     \
      if (resid > 0)                                                    \
        { pos.s[_d]=resid%sSize.s[_d];                                    \
          resid/=sSize.s[_d]; }                                         \
      else                                                              \
          pos.s[_d]=0;                                                    \
  }

// since the c- modula function does not wrap to positive number we define our own modula function
#define MyModulo(x,N) (((x) % (N) + (N)) % (N))

#define IdxNDFromCoords(pos,dSize,idd)                                   \
  (idd)=0;                                                              \
  {                                                                     \
  size_t _Stride=1;                                                \
  for(int _d=0;_d<CUDA_MAXDIM;_d++)                                      \
  if (dSize.s[_d]>0) {                                                   \
              long long N=dSize.s[_d];                                  \
              {(idd) += MyModulo(pos.s[_d],N) *_Stride;}  \
        _Stride *= dSize.s[_d]; }                                        \
}
// This was removed when changed from int to size_t to accomodate 64 bits properly:
// if (pos.s[_d] < 0)                                              
//              {(idd) += (dSize.s[_d]-((-pos.s[_d]) % dSize.s[_d])) *_Stride;}          
//          else                                                          
   
// The macro below converts an ND memory position into a memory position that may have singleton dimensions
// numdims: number of dimensions
// posOrig: original index in ND array (without singleton)
// isSingleton: boolean array denoting whether a dimension needs to be reduced to singleton (size 1)
// stridesOrig: strides of the original array
// posSingleton: resulting Singleton index which can be used
// stridesSingleton: the strides in the result array
// 
// The algorithms goes through all dimensions and allways assumes that the rest of (yet untreated) dimensions is of the same type as the state variable _state: 1 = singleton dimension
#define Original2Singleton(numdims, isSingleton, posOrig, sizesOrig, posSingleton) { \
    posSingleton=posOrig;                                       \
    int _d,_state=0, stridesOrig=1, stridesSingleton=1;         \
    for (_d = 0;_d<numdims;_d++){                               \
        if (_state == 0)    {                                   \
            if (isSingleton.s[_d] != _state)    {               \
                posSingleton = (posSingleton % stridesSingleton);\
                _state = 1;}                                    \
            else stridesSingleton *= sizesOrig.s[_d];                \
        } else {                                                \
            if (isSingleton.s[_d] != _state)     {              \
                posSingleton +=  (posOrig / stridesOrig) * stridesSingleton;      \
            	stridesSingleton *= sizesOrig.s[_d];                \
                _state=0; }                                     \
        }                                                       \
        stridesOrig *= sizesOrig.s[_d];                         \
    }                                                           \
}                                                               \


// The partial reduction function below projects the data along one dimension
// the processors are assigned to the result image pixels
// CAVE: These versions can be slow, if the resulting data has is smaller than the number of processors
#define CUDA_PartRedMask(FktName, OP)               \
__global__ void FktName (float *in, float *out, float * mask, size_t N, size_t ProjStride, size_t ProjSize){      \
  size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  if(idd>=N) return;                                                    \
  size_t p;                                                                \
  size_t ids=((idd%ProjStride) + (idd/ProjStride)*(ProjStride*ProjSize));  \
  ACCUTYPE accu=0.0;                                                       \
  int laterPix=0;                                                       \
  for (p=0;p<ProjSize;p++)                                              \
    {                                                                   \
      if (mask == 0 || mask[ids] != 0.0)                                \
        if (! laterPix)  {                                              \
            accu=in[ids];                                               \
            laterPix=1;                                                 \
        } else {                                                        \
            accu=OP(accu,(ACCUTYPE) in[ids]);                             \
        }                                                               \
      ids += ProjStride;                                                \
    }                                                                   \
 out[idd] = (float) accu;                                               \
}                                                                       \
\
extern "C" const char * CUDA ## FktName(float *a, float * mask, float * c, size_t sSize[CUDA_MAXPROJ], int ProjDir)\
{                                                                       \
    hipError_t myerr;                                                  \
    size_t d,N=1;                                                       \
	size_t blockSize;dim3 nBlocks;                                      \
    size_t ProjStride=1,ProjSize=1;                                     \
    if (ProjDir>CUDA_MAXPROJ)                                           \
        return "Error: Unsupported projection direction";               \
    for (d=0;d<CUDA_MAXPROJ;d++)  {                                     \
        if (d < ProjDir-1)  ProjStride *= sSize[d];                     \
        if (d != ProjDir-1) N*=sSize[d];                                \
    }                                                                   \
    ProjSize=sSize[ProjDir-1];                                          \
    MemoryLayout(N,blockSize,nBlocks)                                   \
	FktName<<<nBlocks,blockSize>>>(a,c,mask,N,ProjStride,ProjSize);     \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}

//   This is the same as the above but suited for complex numbers
#define CUDA_PartRedMaskCpx(FktName, OP)               \
__global__ void FktName (float *in, float *out, float * mask, size_t N, size_t ProjStride, size_t ProjSize){      \
  size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  if(idd>=N) return;                                                    \
  size_t p;                                                                \
  size_t ids=((idd%ProjStride) + (idd/ProjStride)*(ProjStride*ProjSize));  \
  ACCUTYPE accu=0.0;                                                       \
  ACCUTYPE accuI=0.0;                                                      \
  int laterPix=0;                                                       \
  for (p=0;p<ProjSize;p++)                                              \
    {                                                                   \
      if (mask == 0 || mask[ids] != 0.0)                                \
        if (! laterPix)  {                                              \
            accu=in[2*ids];                                             \
            accuI=in[2*ids+1];                                          \
            laterPix=1;                                                 \
        } else {                                                        \
            accu=OP(accu,(ACCUTYPE)in[2*ids]);                           \
            accuI=OP(accuI,(ACCUTYPE)in[2*ids+1]);                       \
        }                                                               \
      ids += ProjStride;                                                \
    }                                                                   \
 out[2*idd] = (float) accu;                                             \
 out[2*idd+1] = (float) accuI;                                          \
}                                                                       \
\
extern "C" const char * CUDA ## FktName(float *a, float * mask, float * c, size_t sSize[3], int ProjDir)\
{                                                                       \
     hipError_t myerr;                                                \
    size_t d,N=1;                                                       \
	size_t blockSize;dim3 nBlocks;                                      \
    size_t ProjStride=1,ProjSize=1;                                     \
    if (ProjDir>CUDA_MAXPROJ)                                           \
        return "Error: Unsupported projection direction";               \
    for (d=0;d<CUDA_MAXPROJ;d++)  {                                     \
        if (d < ProjDir-1)  ProjStride *= sSize[d];                     \
        if (d != ProjDir-1) N*=sSize[d];                                \
    }                                                                   \
    ProjSize=sSize[ProjDir-1];                                          \
    MemoryLayout(N,blockSize,nBlocks)                                   \
	FktName<<<nBlocks,blockSize>>>(a,c,mask,N,ProjStride,ProjSize);     \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}

// This partial reduction code keeps track of the index
#define CUDA_PartRedMaskIdx(FktName, OP)               \
__global__ void FktName (float *in, float *out, float * outIdx, float * mask, size_t N, size_t ProjStride, size_t ProjSize){      \
  size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  if(idd>=N) return;                                                    \
  size_t p;                                                                \
  size_t ids=((idd%ProjStride) + (idd/ProjStride)*(ProjStride*ProjSize));  \
  float accu=0.0;                                                       \
  float accuIdx=-1;                                                     \
  int laterPix=0;                                                       \
  for (p=0;p<ProjSize;p++)                                              \
    {                                                                   \
      if (mask == 0 || mask[ids] != 0.0)                                \
        if (! laterPix)  {                                              \
            accu=in[ids];                                               \
            accuIdx=p;                                                  \
            laterPix=1;                                                 \
        } else {                                                        \
            if (OP(accu,in[ids])) {accu=in[ids];accuIdx=p;}             \
        }                                                               \
      ids += ProjStride;                                                \
    }                                                                   \
 out[idd] = accu;                                                       \
 if (outIdx != 0)                                                       \
    outIdx[idd] = accuIdx;                                                 \
}                                                                       \
\
extern "C" const char * CUDA ## FktName(float *a, float * mask, float * c, float * cIdx, size_t sSize[5], int ProjDir)\
{                                                                       \
    hipError_t myerr;                                                  \
    size_t d,N=1;                                                       \
	size_t blockSize;dim3 nBlocks;                                      \
    size_t ProjStride=1,ProjSize=1;                                     \
    if (ProjDir>CUDA_MAXPROJ)                                           \
        return "Error: Unsupported projection direction";               \
    for (d=0;d<CUDA_MAXPROJ;d++)  {                                     \
        if (d < ProjDir-1)  ProjStride *= sSize[d];                     \
        if (d != ProjDir-1) N*=sSize[d];                                \
    }                                                                   \
    ProjSize=sSize[ProjDir-1];                                          \
    MemoryLayout(N,blockSize,nBlocks)                                   \
	FktName<<<nBlocks,blockSize>>>(a,c,cIdx,mask,N,ProjStride,ProjSize);\
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}

// Below is some reduction code adapted from the tips and tricks tutorial 
// https://www.sharcnet.ca/help/index.php/CUDA_tips_and_tricks
// FUNCTION BELOW IS SLOW AND DOES NOT WORK PROPERLY YET
#define CUDA_FullRedBin(FktName, OP)                                    \
__global__ void FktName (float *in, size_t N){                             \
  const size_t stride = CUIMAGE_REDUCE_THREADS;                    \
  const size_t start  = threadIdx.x;\
  __shared__ float accum[CUIMAGE_REDUCE_THREADS];               \
  ACCUTYPE tmp=0;                                                 \
  size_t nTotalThreads=CUIMAGE_REDUCE_THREADS;                     \
  size_t thread2;                                                  \
                                                                \
  if (start >= CUIMAGE_REDUCE_THREADS) return;                   \
  if (start >= N) {accum[start]=0;return;}                      \
                                                                \
  tmp = in[start];                               \
  for (size_t ii=start+stride; ii < N; ii += CUIMAGE_REDUCE_THREADS)  { \
    tmp = OP(tmp, (ACCUTYPE) in[ii]);        \
  }                                                             \
  accum[threadIdx.x]=tmp;                                       \
  __syncthreads();                                              \
                                                                \
/* Now entering the logaritmic reduction phase of the algorithm*/       \
while(nTotalThreads > 1)                                                \
{                                                                       \
  size_t halfPoint = (nTotalThreads >> 1);	/* divide by two */             \
  /* only the first half of the threads will be active. */              \
                                                                        \
  if (threadIdx.x < halfPoint)                                          \
  {  thread2 = threadIdx.x + halfPoint;                                   \
   /* Skipping the fictious threads blockDim.x ... blockDim_2-1 */      \
   if (thread2 < stride)                                            \
      accum[threadIdx.x]=OP(accum[threadIdx.x],accum[thread2]);         \
  }                                                                     \
  __syncthreads();                                                      \
  /* Reducing the binary tree size by two:  */                          \
  nTotalThreads = halfPoint;                                            \
}                                                                       \
  __syncthreads();                                              \
  if (threadIdx.x == 0)                                         \
          cuda_resultVal.x=accum[0];                            \
}                                                               \
extern "C" const char * CUDA ## FktName(float * a, size_t N, float * resp) \
{                                                               \
  int CUIMAGE_REDUCE_BLOCKS;                                    \
  dim3 threadBlock;                                             \
  dim3 blockGrid;                                               \
  CUIMAGE_REDUCE_BLOCKS=NBLOCKSL(N,CUIMAGE_REDUCE_THREADS);     \
  threadBlock.x=CUIMAGE_REDUCE_THREADS;                         \
  blockGrid.x=CUIMAGE_REDUCE_BLOCKS;                            \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, N);                    \
  if (hipGetLastError() != hipSuccess)                        \
      return hipGetErrorString(hipGetLastError());            \
                                                                \
  hipMemcpyFromSymbol(resp, HIP_SYMBOL(cuda_resultVal), sizeof(* resp));   \
  if (hipGetLastError() != hipSuccess)                        \
      return hipGetErrorString(hipGetLastError());            \
  return 0;                                                     \
}


// Below is the reduction code of Wouter Caarls, modified
// This could potentially also be run sequentially over the remaining dimension

#define CUDA_FullRed(FktName, OP1,OP2)                               \
__global__ void FktName (float *in, ACCU_ARRTYPE *out, size_t N){         \
  const size_t stride = blockDim.x * gridDim.x;                    \
  const size_t start  = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;\
  __shared__ ACCU_ARRTYPE accum[CUIMAGE_REDUCE_THREADS];               \
  ACCUTYPE tmp=0;                                                 \
  if (start >= N) return;                                       \
                                                                \
  tmp = in[start];                                              \
  for (size_t ii=start+stride; ii < N; ii += stride)  {            \
    tmp = OP1(tmp, (ACCUTYPE) in[ii]);                             \
  }                                                             \
  accum[threadIdx.x]=tmp;                                       \
  __syncthreads();                                              \
  if (threadIdx.x == 0)                                         \
  {                                                             \
    ACCUTYPE res = accum[0];                                      \
    size_t limit;                                                  \
    if (start+blockDim.x > N) limit=(N-start);                  \
    else limit=blockDim.x;                                      \
    for (size_t ii = 1; ii < limit; ii++) {                        \
      res=OP2(res,(ACCUTYPE) accum[ii]);                           \
     }                                                          \
    out[blockIdx.x] = res;                                      \
  }                                                             \
}                                                               \
                                                                \
extern "C" const char * CUDA ## FktName(float * a, size_t N, ACCUTYPE * resp) \
{                                                               \
  hipError_t myerr;                                            \
  const char * myerrStr;                                        \
  ACCUTYPE res;                                                    \
  int CUIMAGE_REDUCE_BLOCKS;                                    \
  dim3 threadBlock;                                             \
  dim3 blockGrid;                                               \
  CUIMAGE_REDUCE_BLOCKS=NBLOCKSL(N,CUIMAGE_REDUCE_THREADS);     \
  threadBlock.x=CUIMAGE_REDUCE_THREADS;                         \
  blockGrid.x=CUIMAGE_REDUCE_BLOCKS;                            \
                                                                \
  myerrStr=CheckReduceAllocation(2*CUIMAGE_REDUCE_BLOCKS);      \
  if (myerrStr) return myerrStr;                                \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, TmpRedArray, N);       \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpy(accum, TmpRedArray, CUIMAGE_REDUCE_BLOCKS*sizeof(ACCU_ARRTYPE), hipMemcpyDeviceToHost);\
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  res = (ACCUTYPE) accum[0];                                      \
  for (size_t ii=1; ii < CUIMAGE_REDUCE_BLOCKS; ii++)  {           \
    res=(ACCUTYPE) OP2(res,(ACCUTYPE) accum[ii]);                    \
   }                                                            \
  /* hipFree(TmpRedArray); */                                  \
  /* free(accum); */                                            \
                                                                \
  (* resp)=res;                                                 \
  return 0;                                                     \
}

// The version below is for complex valued arrays

#define CUDA_FullRedCpx(FktName, OP)               \
__global__ void FktName (float *in, ACCU_ARRTYPE *out, size_t N){      \
  const size_t stride = blockDim.x * gridDim.x;                    \
  const size_t start  = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;\
  ACCUTYPE tmpR=0,tmpI=0;                                         \
  __shared__ ACCU_ARRTYPE accum[CUIMAGE_REDUCE_THREADS];               \
  __shared__ ACCU_ARRTYPE accumI[CUIMAGE_REDUCE_THREADS];              \
  if (start >= N) return;                                    \
                                                                \
  tmpR = in[2*start];                             \
  tmpI = in[2*start+1];                          \
  for (size_t ii=start+stride; ii < N; ii += stride)  {         \
    tmpR = OP(tmpR, (ACCUTYPE) in[2*ii]);      \
    tmpI = OP(tmpI, (ACCUTYPE) in[2*ii +1]); \
  }                                                             \
  accum[threadIdx.x]=tmpR;                                       \
  accumI[threadIdx.x]=tmpI;                                     \
  __syncthreads();                                              \
  if (!threadIdx.x)                                             \
  {                                                             \
    ACCUTYPE res = accum[0];                                       \
    ACCUTYPE resI = accumI[0];                                     \
    size_t limit;                                                  \
    if (start+blockDim.x > N) limit=(N-start);  \
    else limit=blockDim.x;                                      \
    for (size_t ii = 1; ii < limit; ii++) {                        \
      res=OP(res,(ACCUTYPE) accum[ii]);                           \
      resI=OP(resI,(ACCUTYPE) accumI[ii]);                        \
     }                                                          \
    out[2*blockIdx.x] = res;                                    \
    out[2*blockIdx.x + 1] = resI;                               \
  }                                                             \
}  \
\
extern "C" const char * CUDA ## FktName(float * a, size_t N, ACCUTYPE * resp) \
{                                                               \
    hipError_t myerr;                                          \
  const char * myerrStr;                                              \
  ACCUTYPE res, resI;                                              \
  int CUIMAGE_REDUCE_BLOCKS;                                    \
  dim3 threadBlock;                                             \
  dim3 blockGrid;                                               \
  CUIMAGE_REDUCE_BLOCKS=NBLOCKSL(N,CUIMAGE_REDUCE_THREADS);     \
  threadBlock.x=CUIMAGE_REDUCE_THREADS;                         \
  blockGrid.x=CUIMAGE_REDUCE_BLOCKS;                            \
                                                                \
  myerrStr=CheckReduceAllocation(2*CUIMAGE_REDUCE_BLOCKS);      \
  if (myerrStr) return myerrStr;                                \
                                                                \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, TmpRedArray, N);       \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpy(accum, TmpRedArray, 2*CUIMAGE_REDUCE_BLOCKS*sizeof(ACCU_ARRTYPE), hipMemcpyDeviceToHost);\
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
                                                                \
  res = (ACCUTYPE) accum[0];                                               \
  resI = (ACCUTYPE) accum[1];                                              \
  for (size_t ii=1; ii < CUIMAGE_REDUCE_BLOCKS; ii++)  {           \
    res=(ACCUTYPE) OP(res,(ACCUTYPE) accum[2*ii]);                                    \
    resI=(ACCUTYPE) OP(resI,(ACCUTYPE) accum[2*ii + 1]);                              \
   }                                                            \
  /* hipFree(interm);  */                                      \
  /* free(accum); */                                            \
                                                                \
  (* resp)=res;                                                 \
  (* (resp+1))=resI;                                            \
  return 0;                                                     \
}

// The version below is for remembering the index (e.g. max and min)

#define CUDA_FullRedIdx(FktName, OP)               \
__global__ void FktName (float *in, ACCU_ARRTYPE *out, size_t size){      \
  const size_t stride = blockDim.x * gridDim.x;                    \
  const size_t start  = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;\
  __shared__ ACCU_ARRTYPE accum[CUIMAGE_REDUCE_THREADS];               \
  __shared__ ACCU_ARRTYPE accumI[CUIMAGE_REDUCE_THREADS];              \
  if (start >= size) return;                                    \
                                                                \
  accum[threadIdx.x] = in[start];                               \
  accumI[threadIdx.x] = start;                                  \
  for (size_t ii=start+stride; ii < size; ii += stride)  {         \
    if OP(accum[threadIdx.x], in[ii]) { accum[threadIdx.x]= in[ii]; accumI[threadIdx.x]= ii; }      \
  }                                                             \
  __syncthreads();                                              \
  if (!threadIdx.x)                                             \
  {                                                             \
    ACCUTYPE res = (ACCUTYPE) accum[0];                         \
    ACCUTYPE resI = (ACCUTYPE) accumI[0];                       \
    size_t limit;                                                  \
    if (start+blockDim.x > size) limit=1+(size-start-1)/gridDim.x;  \
    else limit=blockDim.x;                                      \
    for (size_t ii = 1; ii < limit; ii++) {                        \
    if OP(res, (ACCUTYPE) accum[ii]){ res= (ACCUTYPE) accum[ii]; resI= (ACCUTYPE) accumI[ii]; }  \
     }                                                          \
    out[2*blockIdx.x] = res;                                    \
    out[2*blockIdx.x + 1] = resI;                               \
  }                                                             \
}  \
\
extern "C" const char * CUDA ## FktName(float * a, size_t N, ACCUTYPE * resp) \
{                                                               \
  ACCUTYPE res, resI;                                              \
  hipError_t myerr;                                            \
  const char * myerrStr;                                        \
  int CUIMAGE_REDUCE_BLOCKS;                                    \
  dim3 threadBlock;                                             \
  dim3 blockGrid;                                               \
  CUIMAGE_REDUCE_BLOCKS=NBLOCKSL(N,CUIMAGE_REDUCE_THREADS);     \
  threadBlock.x=CUIMAGE_REDUCE_THREADS;                         \
  blockGrid.x=CUIMAGE_REDUCE_BLOCKS;                            \
                                                                \
  myerrStr=CheckReduceAllocation(2*CUIMAGE_REDUCE_BLOCKS);      \
  if (myerrStr) return myerrStr;                                \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, TmpRedArray, N);       \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpy(accum, TmpRedArray, 2*CUIMAGE_REDUCE_BLOCKS*sizeof(ACCU_ARRTYPE), hipMemcpyDeviceToHost);\
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  res = (ACCUTYPE) accum[0];                                               \
  resI = (ACCUTYPE)accum[1];                                              \
  for (size_t ii=1; ii < CUIMAGE_REDUCE_BLOCKS; ii++)  {           \
    if OP(res, (ACCUTYPE) accum[2*ii]) {res=(ACCUTYPE)accum[2*ii]; resI= (ACCUTYPE) accum[2*ii+1];  }  \
   }                                                            \
  /* hipFree(TmpRedArray); */                                  \
  /* free(accum); */                                            \
                                                                \
  (* resp)=res;                                                 \
  (* (resp+1))=resI;                                            \
  return 0;                                                     \
}


// Allows to work with the linear index image from a binary mask image.
// useful for: a(mask) = 2*a(mask)
// Algorithm: pass1 : count ones in your area
// pass 2: integrate accum over thread number to get block ones offset
// pass 3: Apply index

#define CUDA_MaskIdx(FktName, EXPRESSIONS)                      \
__global__ void FktName (float *a, float * mask,float *c, size_t N){ \
  size_t Blocksize = N/CUIMAGE_REDUCE_THREADS + 1;                 \
  size_t start = Blocksize * threadIdx.x;                          \
  __shared__ size_t accum[CUIMAGE_REDUCE_THREADS+1];               \
  if (start >= N) return;                                       \
                                                                \
  { size_t SumMask=0;                                              \
  for (size_t ii=start; ii < start+Blocksize; ii ++)  {            \
    if (ii < N)                                                 \
        SumMask += (mask[ii] != 0);                             \
  }                                                             \
  accum[threadIdx.x+1] = SumMask;                               \
  }                                                             \
  __syncthreads();                                              \
  if (threadIdx.x == 0)                                         \
  {                                                             \
    accum[0] = 0;                                               \
    size_t res = 0;                                                \
    for (size_t ii = 0; ii*Blocksize < N; ii++) {                  \
      res += accum[ii+1];                                       \
      accum[ii+1] = res;                                        \
     }                                                          \
    cuda_resultInt = res;                                       \
  }                                                             \
  __syncthreads();                                              \
  size_t mask_idx= accum[threadIdx.x];                             \
  for (size_t idx=start; idx < start+Blocksize; idx ++)  {         \
    if ((idx < N) && (mask[idx] != 0))                          \
      {                                                         \
        EXPRESSIONS                                             \
        mask_idx ++;                                            \
      }                                                         \
  }                                                             \
}                                                               \
                                                                \
extern "C" const char * CUDA ## FktName(float * in, float * mask, float *  out, size_t N, size_t * pM) \
{                                                               \
  int CUIMAGE_REDUCE_BLOCKS=1;                                  \
  hipError_t myerr;                                            \
  dim3 threadBlock(CUIMAGE_REDUCE_THREADS);                     \
  dim3 blockGrid(CUIMAGE_REDUCE_BLOCKS);                        \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(in, mask, out, N);        \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpyFromSymbol(pM, HIP_SYMBOL(cuda_resultInt), sizeof(* pM));       \
                                                                \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
  return 0;                                                     \
}

/*  This was for debugging purposes. Commented out for now
#define CUDA_BinaryFktOld(FktName,expression)                          \
__global__ void                                                     \
FktName(float*a,float *b, float * c, size_t N)                         \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
	if(idx>=N) return;                                              \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float * b, float * c, size_t N, int numdims, SizeND sizesC, BoolND isSingletonA, BoolND isSingletonB)  \
{                                                                       \
    hipError_t myerr;                                          \
	size_t blockSize;dim3 nBlocks;                                         \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,b,c,N);                            \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       
*/

// In the expression one can use the variables idx (for real valued arrays) and idc (for complex valued arrays)
// -------------- caller function is also generated -------------
// 
// The 2 macros below treat binary functions such as plus (as the one before)
// but singleton dimensions will be wrapped just like in Python or DIPImage
#define CUDA_BinaryFkt(FktName,expression)                          \
__global__ void                                                     \
FktName(float*a,float *b, float * c, size_t N)                         \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
    size_t idxA=idx, idxB=idx;                                         \
	if(idx>=N) return;                                              \
	expression                                                      \
}                                                                   \
__global__ void                                                     \
FktName ##_S(float*a,float *b, float * c, size_t N, int numdims, SizeND sizesC, BoolND isSingletonA, BoolND isSingletonB) \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
    size_t idxA,idxB;                                                  \
	if(idx>=N) return;                                              \
    Original2Singleton(numdims, isSingletonA, idx,sizesC,idxA)     \
    Original2Singleton(numdims, isSingletonB, idx,sizesC,idxB)     \
    expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float * b, float * c, size_t N, int numdims, SizeND sizesC, BoolND isSingletonA, BoolND isSingletonB)  \
{                                                                       \
    hipError_t myerr;                                                  \
	size_t blockSize;dim3 nBlocks;                                         \
    myerr=hipGetLastError();                                           \
    if (numdims==0) {                                                            \
    MemoryLayout(N,blockSize,nBlocks)                                   \
	FktName<<<nBlocks,blockSize>>>(a,b,c,N);                            \
    } else                                                              \
    {                                                                   \
    MemoryLayout(N,blockSize,nBlocks)                                   \
	FktName ## _S<<<nBlocks,blockSize>>>(a,b,c,N, numdims, sizesC, isSingletonA, isSingletonB);  \
    }                                                                   \
    myerr=hipGetLastError();                                           \
    if (myerr != hipSuccess)                                           \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

// In the expression one can use the variables idx (for real valued arrays) and idc (for complex valued arrays)
// -------------- caller function is also generated -------------
// 
// The 2 macros below treat functions with an arbitrary number of reference arrays
// but singleton dimensions will be wrapped just like in Python or DIPImage

#define CUDA_NArgsFkt(FktName,expression,NArgs)                     \
typedef struct {                                                    \
    float * s[NArgs];                                                  \
} FktName ##_ARGTYPE ;                                             \
__global__ void                                                     \
FktName(FktName ##_ARGTYPE f,float * c, size_t N)                         \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
    size_t idxArg[NArgs],myarg;                                        \
	if(idx>=N) return;                                              \
     for (myarg=0;myarg<NArgs;myarg++)                              \
        idxArg[myarg]=idx;                                          \
	expression                                                      \
}                                                                   \
__global__ void                                                     \
FktName ##_S(FktName ##_ARGTYPE f,float * c, size_t N, int numdims, SizeND sizesC, BoolND isSingleton[NArgs]) \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
    size_t idxArg[NArgs],myarg;                                              \
	if(idx>=N) return;                                              \
    for (myarg=0;myarg<NArgs;myarg++)                               \
        {Original2Singleton(numdims, isSingleton[NArgs], idx,sizesC,idxArg[myarg]) }\
    expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * f[NArgs], float * c, size_t N, int numdims, SizeND sizesC, BoolND isSingleton[NArgs])  \
{                                                                       \
    hipError_t myerr;                                                  \
	size_t blockSize,n;dim3 nBlocks;                                         \
    FktName ##_ARGTYPE F;                                               \
    for (n=0;n<NArgs;n++) F.s[n]=f[n];                                  \
    myerr=hipGetLastError();                                           \
    if (numdims==0) {                                                   \
    MemoryLayout(N,blockSize,nBlocks)                                   \
	FktName<<<nBlocks,blockSize>>>(F,c,N);                            \
    } else                                                              \
    {                                                                   \
    MemoryLayout(N,blockSize,nBlocks)                                   \
	FktName ## _S<<<nBlocks,blockSize>>>(F,c,N, numdims, sizesC, isSingleton);  \
    }                                                                   \
    myerr=hipGetLastError();                                           \
    if (myerr != hipSuccess)                                           \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

// In the expression one can use the variables idx (for real valued arrays) 
// -------------- caller function is also generated -------------
#define CUDA_IndexFkt(FktName,expression)                          \
__global__ void                                                     \
FktName(float*a,float *b, float * c, size_t N, size_t M)                         \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
	if(idx>=M) return;                                              \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float * b, float * c, size_t N, size_t M)  \
{                                                                       \
    hipError_t myerr;                                          \
	size_t blockSize;dim3 nBlocks;                                         \
    MemoryLayout(M,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,b,c,N,M);                            \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

// --------------Macro generating operation of array with real constant -------------

#define CUDA_UnaryFktConst(FktName,expression)                      \
__global__ void                                                     \
FktName(float*a,float b, float * c, size_t N)                          \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
	if(idx>=N) return;                                              \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float b, float * c, size_t N)  \
{                                                                       \
    hipError_t myerr;                                          \
	size_t blockSize;dim3 nBlocks;                                         \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,b,c,N);                            \
    myerr=hipGetLastError();                                             \
    if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

// --------------Macro generating operation with complex array and constant -------------
#define CUDA_UnaryFktConstC(FktName,expression)                      \
__global__ void                                                     \
FktName(float*a,float br, float bi, float * c, size_t N)               \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
	if(idx>=N) return;                                              \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float br, float bi, float * c, size_t N)  \
{                                                                       \
    hipError_t myerr;                                                  \
	size_t blockSize;dim3 nBlocks;                                         \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,br,bi,c,N);                        \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

        

// ----------- Makro for function with an integer Vector ---- e.g.- for cyclic shifts etc. -----
#define CUDA_UnaryFktIntVec(FktName,expression)                      \
__global__ void                                                     \
FktName(float*a, SizeND b, float * c, SizeND sSize, size_t N)          \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
	if(idx>=N) return;                                              \
    expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, size_t b[CUDA_MAXDIM], float * c, size_t mySize[CUDA_MAXDIM], size_t N)  \
{                                                                       \
  hipError_t myerr;                                          \
	size_t blockSize;dim3 nBlocks;                                         \
    SizeND sb,sSize;                                                    \
    for (size_t d=0;d<CUDA_MAXDIM;d++)                                     \
    { sb.s[d]=b[d];sSize.s[d]=mySize[d]; }                              \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,sb,c,sSize,N);                     \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

// ----------- Makro for function with an integer Vector ---- e.g.- for cyclic shifts etc. -----
#define CUDA_Fkt2Vec(FktName,expression)                            \
__global__ void                                                     \
FktName(float * c, VecND vec1, VecND vec2, SizeND sSize, size_t N)     \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
	if(idx>=N) return;                                              \
    expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * c, VecND vec1, VecND vec2, SizeND sSize, size_t N)  \
{                                                                       \
    hipError_t myerr;                                          \
	size_t blockSize;dim3 nBlocks;                                         \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(c,vec1,vec2,sSize,N);                \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}



// --------------Macro generating unary operation with complex array  -------------
#define CUDA_UnaryFkt(FktName,expression)                     \
__global__ void                                                     \
FktName(float*a, float * c, size_t N)                                  \
{                                                                   \
    size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
	if(idx>=N) return;                                              \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float * c, size_t N)         \
{                                                                       \
    hipError_t myerr;                                          \
	size_t blockSize;dim3 nBlocks;                                         \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,c,N);                              \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

// ---------------------- Some functions which know about x, and z position --------
// gets two sources and one destination, the two sources are assumed to have the same size
// sx,sy,sz : Source array sizes (total)
// sox,soy,soy : offsets
// ssx, ssy,ssz : source (or destination) subarray sizes
// dx,dy,dz: destination total array sizes
// dox,doy,doz : destination offsets


// Line below is used as an add-on to the 3d function below in case 3d assignment is needed
#define GET3DIDD size_t idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]);

#define CUDA_3DFkt(FktName,expressions)                                  \
__global__ void                                                         \
FktName(float *a, float *c, Size3D sSize,Size3D dSize,Size3D sOffs, Size3D sROI, Size3D dOffs) \
{                                                                       \
  size_t N=sROI.s[0]*sROI.s[1]*sROI.s[2];                                        \
  size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  size_t x=(idx)%sROI.s[0];                                                  \
  size_t y=(idx/sROI.s[0])%sROI.s[1];                                          \
  size_t z=(idx/(sROI.s[0]*sROI.s[1]))%sROI.s[2];                                \
  size_t ids=x+sOffs.s[0]+sSize.s[0]*(y+sOffs.s[1])+sSize.s[0]*sSize.s[1]*(z+sOffs.s[2]);                               \
  if(idx>=N) return;                                              \
  expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * a, float *c, size_t sSize[3], size_t dSize[3], size_t sOffs[3], size_t sROI[3], long long dOffs[3])  \
{                                                                       \
    hipError_t myerr;                                                \
    size_t N=sROI[0]*sROI[1]*sROI[2];                                      \
	size_t blockSize;dim3 nBlocks;                                         \
     Size3D sS,dS,sO,sR,dO;                                              \
    int d;                                                              \
    for (d=0;d<3;d++)                                                   \
        {sS.s[d]=sSize[d];dS.s[d]=dSize[d];sO.s[d]=sOffs[d];sR.s[d]=sROI[d];dO.s[d]=dOffs[d];} \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,c,sS,dS,sO,sR,dO); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                     

// --- macros for sub-assigning a block with vectors in each dimension -----
#define CUDA_3DAsgFkt(FktName,expressions)                                  \
__global__ void                                                         \
FktName(float *c, float br, float bi, Size3D dSize, Size3D dROI, Size3D dOffs) \
{                                                                       \
  size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  size_t N=dROI.s[0]*dROI.s[1]*dROI.s[2];                                        \
  if(idx>=N) return;                                                    \
  size_t x=(idx)%dROI.s[0];                                               \
  size_t y=(idx/dROI.s[0])%dROI.s[1];                                    \
  size_t z=(idx/(dROI.s[0]*dROI.s[1]))%dROI.s[2];                       \
  size_t idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]);                               \
  expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * c, float br, float bi, size_t dSize[3], size_t dROI[3], size_t dOffs[3])  \
{                                                                       \
    hipError_t myerr;                                                \
    size_t N=dROI[0]*dROI[1]*dROI[2];                                      \
	size_t blockSize;dim3 nBlocks;                                         \
    Size3D dR,dS,dO;                                              \
    int d;                                                              \
    for (d=0;d<3;d++)                                                   \
        {dS.s[d]=dSize[d];dR.s[d]=dROI[d];dO.s[d]=dOffs[d];} \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(c,br,bi,dS,dR,dO); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                     

// --- macros for sub-assigning a block with vectors in each dimension - Extended version to be suitable for repmat
#define CUDA_3DWrapAsgFkt(FktName,expressions)                          \
__global__ void                                                         \
FktName(float *a, float *c, Size3D dSize, Size3D sSize)       \
{                                                                       \
  size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  size_t N=dSize.s[0]*dSize.s[1]*dSize.s[2];                               \
  size_t x=(idd)%dSize.s[0];                                                \
  size_t y=(idd/dSize.s[0])%dSize.s[1];                                    \
  size_t z=(idd/(dSize.s[0]*dSize.s[1]))%dSize.s[2];                       \
  size_t ids=x%sSize.s[0]+sSize.s[0]*(y%sSize.s[1])+sSize.s[0]*sSize.s[1]*(z%sSize.s[2]); \
  if(idd>=N) return;                                                    \
  expressions                                                           \
}                                                                       \
extern "C" const char * CUDA ## FktName(float *a, float * c, size_t sSize[3], size_t dSize[3])  \
{                                                                       \
    hipError_t myerr;                                                \
    size_t N=dSize[0]*dSize[1]*dSize[2];                                      \
	size_t blockSize;dim3 nBlocks;                                         \
    int d;                                                              \
    Size3D sS,dS;                                              \
    for (d=0;d<3;d++)                                                   \
        {dS.s[d]=dSize[d];sS.s[d]=sSize[d];} \
   MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,c,dS,sS); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

//  Now the 5D Versions of the same code


#define GETXYZET(aSize,idx)                                             \
  size_t x=(idx)%aSize.s[0];                                          \
  size_t y=(idx/aSize.s[0])%aSize.s[1];                                    \
  size_t z=(idx/(aSize.s[0]*aSize.s[1]))%aSize.s[2];                       \
  size_t t=(idx/(aSize.s[0]*aSize.s[1]*aSize.s[2]))%aSize.s[3];            \
  size_t e=(idx/(aSize.s[0]*aSize.s[1]*aSize.s[2]*aSize.s[3]))%aSize.s[4]; \

#define GET5DIDS size_t ids=x*sStep.s[0]+sOffs.s[0]+sSize.s[0]*(y*sStep.s[1]+sOffs.s[1])+sSize.s[0]*sSize.s[1]*(z*sStep.s[2]+sOffs.s[2])+sSize.s[0]*sSize.s[1]*sSize.s[2]*(t*sStep.s[3]+sOffs.s[3])+sSize.s[0]*sSize.s[1]*sSize.s[2]*sSize.s[3]*(e*sStep.s[4]+sOffs.s[4]);   \

// Line below is used as an add-on to the 5d function below in case 5d assignment is needed
#define GET5DIDD_STEP size_t idd=x*dStep.s[0]+dOffs.s[0]+dSize.s[0]*(y*dStep.s[1]+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z*dStep.s[2]+dOffs.s[2])+dSize.s[0]*dSize.s[1]*dSize.s[2]*(t*dStep.s[3]+dOffs.s[3])+dSize.s[0]*dSize.s[1]*dSize.s[2]*dSize.s[3]*(e*dStep.s[4]+dOffs.s[4]);

#define GET5DIDD size_t idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])+dSize.s[0]*dSize.s[1]*dSize.s[2]*(t+dOffs.s[3])+dSize.s[0]*dSize.s[1]*dSize.s[2]*dSize.s[3]*(e+dOffs.s[4]);

#define CUDA_5DFkt(FktName,expressions)                                 \
__global__ void                                                         \
FktName(float *a, float *c, Size5D sSize,Size5D dSize,Size5D sOffs, Size5D sROI, Size5D dOffs, Size5D sStep, Size5D dStep) \
{                                                                     \
  size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  size_t N=sROI.s[0]*sROI.s[1]*sROI.s[2]*sROI.s[3]*sROI.s[4];            \
  GETXYZET(sROI,idx)                                                      \
  GET5DIDS;                                                               \
  if(idx>=N) return;                                                  \
  expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * a, float *c, Size5D sSize, Size5D dSize, Size5D sOffs, Size5D sROI, Size5D dOffs, Size5D sStep, Size5D dStep)  \
{                                                                       \
    hipError_t myerr;                                                \
    size_t N=sROI.s[0]*sROI.s[1]*sROI.s[2]*sROI.s[3]*sROI.s[4];                      \
	size_t blockSize;dim3 nBlocks;                                         \
    MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,c,sSize,dSize,sOffs,sROI,dOffs,sStep,dStep); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      
                
// --- macros for sub-assigning a block with vectors in each dimension -----

#define CUDA_5DAsgFkt(FktName,expressions)                                  \
__global__ void                                                         \
FktName(float *c, float br, float bi, Size5D dSize, Size5D dROI, Size5D dOffs, Size5D dStep) \
{                                                                       \
  size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  size_t N=dROI.s[0]*dROI.s[1]*dROI.s[2]*dROI.s[3]*dROI.s[4];              \
  GETXYZET(dROI,idx)                                                      \
  GET5DIDD_STEP                                                                \
  if(idx>=N) return;                                                    \
   expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * c, float br, float bi, Size5D dSize, Size5D dROI, Size5D dOffs, Size5D dStep)  \
{                                                                       \
    hipError_t myerr;                                                \
    size_t N=dROI.s[0]*dROI.s[1]*dROI.s[2]*dROI.s[3]*dROI.s[4];            \
	size_t blockSize;dim3 nBlocks;                                         \
   MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(c,br,bi,dSize,dROI,dOffs,dStep);       \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

// --- macros for sub-assigning a block with vectors in each dimension - Extended version to be suitable for repmat
#define CUDA_5DWrapAsgFkt(FktName,expressions)                          \
__global__ void                                                         \
FktName(float *a, float *c, Size5D dSize, Size5D sSize)       \
{                                                                       \
  size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);  \
  size_t N=dSize.s[0]*dSize.s[1]*dSize.s[2]*dSize.s[3]*dSize.s[4];         \
  GETXYZET(dSize,idd)                                                   \
  size_t ids=x%sSize.s[0]+sSize.s[0]*(y%sSize.s[1])+sSize.s[0]*sSize.s[1]*(z%sSize.s[2]) + sSize.s[0]*sSize.s[1]*sSize.s[2]*(t%sSize.s[3])+sSize.s[0]*sSize.s[1]*sSize.s[2]*sSize.s[3]*(e%sSize.s[4]); \
  if(idd>=N) return;                                                    \
  expressions                                                           \
}                                                                       \
extern "C" const char * CUDA ## FktName(float *a, float * c, size_t sSize[5], size_t dSize[5])  \
{                                                                       \
    hipError_t myerr;                                                  \
    size_t N=dSize[0]*dSize[1]*dSize[2]*dSize[3]*dSize[4];                 \
	size_t blockSize;dim3 nBlocks;                                         \
    Size5D sS,dS;                                                       \
    int d;                                                              \
    for (d=0;d<5;d++)                                                   \
        {dS.s[d]=dSize[d];sS.s[d]=sSize[d];}                            \
   MemoryLayout(N,blockSize,nBlocks)                                     \
	FktName<<<nBlocks,blockSize>>>(a,c,dS,sS);                          \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      
        
// The function below checks whether the size of allocated reduce arrays is sufficient and reallocates if needed be
// The arrays are "accum" and "TmpRedArray"
const char * CheckReduceAllocation(size_t asize) {
    hipError_t myerr;
    asize=((asize/MinRedBlockSize) + 1)*MinRedBlockSize;  // round it up to the nearest multiple of MinRedSize
    if (! accum){
       accum = (ACCU_ARRTYPE *) malloc(asize*sizeof(ACCU_ARRTYPE));
       if (! accum)
       return "CheckReduceAllocation: Malloc failed";
    }    
    if (! TmpRedArray) {
        hipMalloc((void **) &TmpRedArray, asize*sizeof(ACCU_ARRTYPE));
        CurrentRedSize=asize;
        myerr=hipGetLastError();
        if (myerr != hipSuccess)
          return hipGetErrorString(myerr);
    }
    
    if (asize > CurrentRedSize)
    {
        free(accum);
        accum = (ACCU_ARRTYPE *) malloc(asize*sizeof(ACCU_ARRTYPE));
        if (! accum)
            return "CheckReduceAllocation: ReMalloc failed";
        hipFree(TmpRedArray);
        myerr=hipGetLastError();
        if (myerr != hipSuccess)
            return hipGetErrorString(myerr);

        hipMalloc((void **) &TmpRedArray, asize*sizeof(ACCU_ARRTYPE));
        myerr=hipGetLastError();
        if (myerr != hipSuccess)
            return hipGetErrorString(myerr);
        CurrentRedSize=asize;
    }
    return 0;
}

extern "C" size_t GetCurrentRedSize(void) {
    return CurrentRedSize;
}

int GetMaxThreads(void) {
    return prop.maxThreadsPerBlock;
}

long GetMaxBlocksX(void) {
    // return min(prop.maxGridSize[0],65535);  // Why does it not work, if this is bigger than 65535 ??
    return prop.maxGridSize[0];  // Why does it not work, if this is bigger than 65535 ??
}

hipDeviceProp_t GetDeviceProp(void) {
    return prop;
}

/*__global__ void                                                         \
bla_ ## FktName(float*a, float * c, int N,  Size3D sSize,Size3D dSize,Size3D sOffs, Size3D sROI, Size3D dOffs) {                                    \
  int idx=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  int idcd=0,idcs=0,ids=0;                                                    \
  if(idx>=N) return;                                                    \
    expression                                                          \
}   \  */

//	FktName<<<nBlocks,blockSize>>>(a,c,sSize,dSize,sOffs, sROI, dOffs); \


CUDA_FullRed(sumpos_arr,mysumpos,mysum)  // only sums over the number of positive values
CUDA_FullRed(sum_arr,mysum,mysum)
//CUDA_FullRedBin(sum_arr,mysum)
CUDA_FullRedCpx(sum_carr,mysum)
// CUDA_FullRed(sum_carr,res+=accum[ii];)
CUDA_FullRedIdx(max_arr,maxCond)
CUDA_FullRedIdx(min_arr,minCond)

CUDA_PartRedMask(psum_arr,mysum)
CUDA_PartRedMaskCpx(psum_carr,mysum)
CUDA_PartRedMaskIdx(pmax_arr,maxCond)
CUDA_PartRedMaskIdx(pmin_arr,minCond)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// CUDA_3DFkt(arr_subcpy_arr,c[idd]=a[ids];)
CUDA_3DAsgFkt(const_3dsubcpy_arr,c[idd]=br;)
CUDA_3DAsgFkt(cconst_3dsubcpy_carr,c[2*idd]=br;c[2*idd+1]=bi;)

// repcopy for repmat command
CUDA_3DWrapAsgFkt(arr_3drepcpy_arr,c[idd]=a[ids];)
CUDA_3DWrapAsgFkt(crepcpy_carr,c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign

CUDA_5DAsgFkt(const_5dsubcpy_arr,c[idd]=br;)
CUDA_5DAsgFkt(cconst_5dsubcpy_carr,c[2*idd]=br;c[2*idd+1]=bi;)

// repcopy for repmat command
CUDA_5DWrapAsgFkt(arr_5drepcpy_arr,c[idd]=a[ids];)
CUDA_5DWrapAsgFkt(carr_5drepcpy_carr,c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)

// Assigning constant values to arrays accessed with a boolean array
CUDA_UnaryFktConst(arr_boolassign_const,if (a[idx]!=0) c[idx]=b;)

CUDA_UnaryFktConstC(carr_boolassign_const,if (a[idx]!=0) {c[2*idx]=br;c[2*idx+1]=bi;})

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// CUDA_3DFkt(arr_subcpy_arr,c[idd]=a[ids];)
CUDA_3DFkt(subcpy_arr, GET3DIDD; c[idd]=a[ids];)
CUDA_3DFkt(carr_3dsubcpy_carr, GET3DIDD; c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)
CUDA_3DFkt(arr_3dsubcpy_carr, GET3DIDD; c[2*idd]=a[ids];c[2*idd+1]=0;)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// These versions intoduce a transpose operation
CUDA_3DFkt(arr_3dsubcpyT_arr,  size_t iddt=y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]); c[iddt]=a[ids];)
CUDA_3DFkt(carr_3dsubcpyT_carr,size_t idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=a[2*ids+1];)
// with conjugation
CUDA_3DFkt(carr_3dsubcpyCT_carr,size_t idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=-a[2*ids+1];)

//CUDA_3DFkt(arr_subref_arr3d,c[idd]=)
//getCudaRef(prhs[1]),newarr,sSize,dSize,cuda_array[newref[0]],cuda_array[newref[1]],cuda_array[newref[2]]);

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
CUDA_5DFkt(arr_5dsubcpy_arr, GET5DIDD_STEP; c[idd]=a[ids];)
CUDA_5DFkt(carr_5dsubcpy_carr, GET5DIDD_STEP; c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)
CUDA_5DFkt(arr_5dsubcpy_carr, GET5DIDD_STEP; c[2*idd]=a[ids];c[2*idd+1]=0;)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// These versions intoduce a transpose operation
CUDA_5DFkt(arr_5dsubcpyT_arr,  size_t iddt=y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]); c[iddt]=a[ids];)
CUDA_5DFkt(carr_5dsubcpyT_carr,size_t idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=a[2*ids+1];)
 // with conjugation
CUDA_5DFkt(carr_5dsubcpyCT_carr,size_t idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=-a[2*ids+1];)  


// Power
CUDA_BinaryFkt(arr_power_arr,c[idx]=pow(a[idxA],b[idxB]);)
CUDA_UnaryFktConst(arr_power_const,c[idx]=pow(a[idx],b);)
CUDA_UnaryFktConst(const_power_arr,c[idx]=pow(b,a[idx]);)

// Multiplications
CUDA_BinaryFkt(arr_times_arr,c[idx]=a[idxA]*b[idxB];)
CUDA_BinaryFkt(carr_times_carr,
    size_t idc=2*idx;size_t idcA=2*idxA;size_t idcB=2*idxB;
    float myr=a[idcA]*b[idcB]-a[idcA+1]*b[idcB+1];float myi=a[idcA]*b[idcB+1]+a[idcA+1]*b[idcB];
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_BinaryFkt(arr_times_carr,size_t idc=2*idx;size_t idcB=2*idxB;c[idc]=a[idxA]*b[idcB];c[idc+1]=a[idxA]*b[idcB+1];)
CUDA_BinaryFkt(carr_times_arr,size_t idc=2*idx;size_t idcA=2*idxA;c[idc]=a[idcA]*b[idxB];c[idc+1]=a[idcA+1]*b[idxB];)
//CUDA_BinaryFkt(arr_times_carr,c[2*idx]=a[idx]*b[2*idx];c[2*idx+1]=a[idx+1]*b[2*idx];)
CUDA_UnaryFktConst(arr_times_const,c[idx]=a[idx]*b;)
CUDA_UnaryFktConst(const_times_arr,c[idx]=a[idx]*b;)
CUDA_UnaryFktConstC(carr_times_const,
    size_t idc=2*idx;
    float myr=a[idc]*br-a[idc+1]*bi;float myi=a[idc]*bi+a[idc+1]*br;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(const_times_carr,
    size_t idc=2*idx;
    float myr=a[idc]*br-a[idc+1]*bi;float myi=a[idc]*bi+a[idc+1]*br;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(arr_times_Cconst,c[2*idx]=a[idx]*br;c[2*idx+1]=a[idx]*bi;)
CUDA_UnaryFktConstC(Cconst_times_arr,c[2*idx]=br*a[idx];c[2*idx+1]=bi*a[idx];)

// Divisions
CUDA_BinaryFkt(arr_divide_arr,c[idx]=a[idxA]/b[idxB];)
CUDA_BinaryFkt(carr_divide_carr,
    size_t idc=2*idx;size_t idcA=2*idxA;size_t idcB=2*idxB;
    float tmp=b[idcB]*b[idcB]+b[idcB+1]*b[idcB+1];
    float myr=(a[idcA]*b[idcB]+a[idcA+1]*b[idcB+1])/tmp;float myi=(a[idcA+1]*b[idcB]-a[idcA]*b[idcB+1])/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_BinaryFkt(carr_divide_arr,size_t idc=2*idx;size_t idcA=2*idxA; c[idc]=a[idcA]/b[idxB];c[idc+1]=a[idcA+1]/b[idxB];)
CUDA_BinaryFkt(arr_divide_carr,
    size_t idc=2*idx;size_t idcB=2*idxB;
    float tmp=b[idcB]*b[idcB]+b[idcB+1]*b[idcB+1];
    float myr=(a[idxA]*b[idcB]+a[idxA+1]*b[idcB+1])/tmp;float myi=(a[idxA+1]*b[idcB]-a[idxA]*b[idcB+1])/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConst(arr_divide_const,c[idx]=a[idx]/b;)
CUDA_UnaryFktConst(const_divide_arr,c[idx]=b/a[idx];)
CUDA_UnaryFktConstC(carr_divide_const,
    size_t idc=2*idx;
    float tmp=br*br+bi*bi;
    float myr=(a[idc]*br+a[idc+1]*bi)/tmp;float myi=(a[idc+1]*br-a[idc]*bi)/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(const_divide_carr,
    size_t idc=2*idx;
    float tmp=a[idc]*a[idc]+a[idc+1]*a[idc+1];
    float myr=(br*a[idc]+bi*a[idc+1])/tmp;float myi=(bi*a[idc]-br*a[idc+1])/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(arr_divide_Cconst,
    float tmp=br*br+bi*bi;
    float myr=a[idx]*br/tmp;float myi= -a[idx]*bi/tmp;
    c[2*idx]=myr;c[2*idx+1]=myi;
)
CUDA_UnaryFktConstC(Cconst_divide_arr,c[2*idx]=br/a[idx];c[2*idx+1]=bi/a[idx];)

// Element-wise maximum operations
CUDA_BinaryFkt(arr_max_arr,c[idx]=a[idxA]>b[idxB]?a[idxA]:b[idxB];)
CUDA_BinaryFkt(carr_max_carr, size_t idc=2*idx;size_t idcA=2*idxA;size_t idcB=2*idxB; if (a[idcA]*a[idcA]+a[idcA+1]*a[idcA+1] > b[idcB]*b[idcB]+b[idcB+1]*b[idcB+1]) {c[idc]=a[idcA];c[idc+1]=a[idcA+1];}else{ c[idc]=b[idcB];c[idc+1]=b[idcB+1];})
CUDA_BinaryFkt(carr_max_arr,size_t idc=2*idx;size_t idcA=2*idxA; if (a[idcA]*a[idcA]+a[idcA+1]*a[idcA+1] > b[idxB]*b[idxB]) {c[idc]=a[idcA];c[idc+1]=a[idcA+1];}else{ c[idc]=b[idxB];c[idc+1]=0;})
CUDA_BinaryFkt(arr_max_carr,size_t idc=2*idx;size_t idcB=2*idxB; if (a[idxA]*a[idxA] > b[idcB]*b[idcB]+b[idcB+1]*b[idcB+1]) {c[idc]=a[idxA];c[idc+1]=0;}else{ c[idc]=b[idcB];c[idc+1]=b[idcB+1];})
CUDA_UnaryFktConst(arr_max_const,c[idx]=a[idx]>b?a[idx]:b;)
CUDA_UnaryFktConst(const_max_arr,c[idx]=a[idx]>b?a[idx]:b;)
CUDA_UnaryFktConstC(carr_max_const,size_t idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] > br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(const_max_carr,size_t idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] > br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(arr_max_Cconst,size_t idc=2*idx;if (a[idx]*a[idx] > br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(Cconst_max_arr,size_t idc=2*idx;if (a[idx]*a[idx] > br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})

// Element-wise minimum operations
CUDA_BinaryFkt(arr_min_arr,c[idx]=a[idxA]<b[idxB]?a[idxA]:b[idxB];)
CUDA_BinaryFkt(carr_min_carr, size_t idc=2*idx;size_t idcA=2*idxA;size_t idcB=2*idxB; if (a[idcA]*a[idcA]+a[idcA+1]*a[idcA+1] < b[idcB]*b[idcB]+b[idcB+1]*b[idcB+1]) {c[idc]=a[idcA];c[idc+1]=a[idcA+1];}else{ c[idc]=b[idcB];c[idc+1]=b[idcB+1];})
CUDA_BinaryFkt(carr_min_arr,size_t idc=2*idx;size_t idcA=2*idxA; if (a[idcA]*a[idcA]+a[idcA+1]*a[idcA+1] < b[idxB]*b[idxB]) {c[idc]=a[idcA];c[idc+1]=a[idcA+1];}else{ c[idc]=b[idxB];c[idc+1]=0;})
CUDA_BinaryFkt(arr_min_carr,size_t idc=2*idx;size_t idcB=2*idxB; if (a[idxA]*a[idxA] < b[idcB]*b[idcB]+b[idcB+1]*b[idcB+1]) {c[idc]=a[idxA];c[idc+1]=0;}else{ c[idc]=b[idcB];c[idc+1]=b[idcB+1];})
CUDA_UnaryFktConst(arr_min_const,c[idx]=a[idx]<b?a[idx]:b;)
CUDA_UnaryFktConst(const_min_arr,c[idx]=a[idx]<b?a[idx]:b;)
CUDA_UnaryFktConstC(carr_min_const,size_t idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] < br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(const_min_carr,size_t idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] < br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(arr_min_Cconst,size_t idc=2*idx;if (a[idx]*a[idx] < br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(Cconst_min_arr,size_t idc=2*idx;if (a[idx]*a[idx] < br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})

// Additions
CUDA_BinaryFkt(arr_plus_arr,c[idx]=a[idxA]+b[idxB];)
CUDA_BinaryFkt(carr_plus_carr, size_t idc=2*idx;size_t idcA=2*idxA;size_t idcB=2*idxB; c[idc]=a[idcA]+b[idcB];c[idc+1]=a[idcA+1]+b[idcB+1];)
CUDA_BinaryFkt(carr_plus_arr,size_t idc=2*idx;size_t idcA=2*idxA;c[idc]=a[idcA]+b[idxB];c[idc+1]=a[idcA+1];)
CUDA_BinaryFkt(arr_plus_carr,size_t idc=2*idx;size_t idcB=2*idxB;c[idc]=a[idxA]+b[idcB];c[idc+1]=b[idcB+1];)
CUDA_UnaryFktConst(arr_plus_const,c[idx]=a[idx]+b;)
CUDA_UnaryFktConst(const_plus_arr,c[idx]=a[idx]+b;)
CUDA_UnaryFktConstC(carr_plus_const,size_t idc=2*idx;c[idc]=a[idc]+br;c[idc+1]=a[idc+1]+bi;)
CUDA_UnaryFktConstC(const_plus_carr,size_t idc=2*idx;c[idc]=a[idc]+br;c[idc+1]=a[idc+1]+bi;)
CUDA_UnaryFktConstC(arr_plus_Cconst,size_t idc=2*idx;c[idc]=a[idx]+br;c[idc+1]=bi;)
CUDA_UnaryFktConstC(Cconst_plus_arr,size_t idc=2*idx;c[idc]=br+a[idx];c[idc+1]=bi;)

// Subtractions
CUDA_BinaryFkt(arr_minus_arr,c[idx]=a[idxA]-b[idxB];)
CUDA_BinaryFkt(carr_minus_carr,size_t idc=2*idx;size_t idcA=2*idxA;size_t idcB=2*idxB; c[idc]=a[idcA]-b[idcB];c[idc+1]=a[idcA+1]-b[idcB+1];)
CUDA_BinaryFkt(carr_minus_arr,size_t idc=2*idx;size_t idcA=2*idxA;c[idc]=a[idcA]-b[idxB];c[idc+1]=a[idcA+1];)
CUDA_BinaryFkt(arr_minus_carr,size_t idc=2*idx;size_t idcB=2*idxB;c[idc]=a[idxA]-b[idcB];c[idc+1]=-b[idcB+1];)
CUDA_UnaryFktConst(arr_minus_const,c[idx]=a[idx]-b;)
CUDA_UnaryFktConst(const_minus_arr,c[idx]=b-a[idx];)
CUDA_UnaryFktConstC(carr_minus_const,size_t idc=2*idx;c[idc]=a[idc]-br;c[idc+1]=a[idc+1]-bi;)
CUDA_UnaryFktConstC(const_minus_carr,size_t idc=2*idx;c[idc]=br-a[idc];c[idc+1]=bi-a[idc+1];)
CUDA_UnaryFktConstC(arr_minus_Cconst,size_t idc=2*idx;c[idc]=a[idx]-br;c[idc+1]=-bi;)
CUDA_UnaryFktConstC(Cconst_minus_arr,size_t idc=2*idx;c[idc]=br-a[idx];c[idc+1]=bi;)

// Referencing and assignment  // STILL NEEDS SOME WORK
// CUDA_BinaryFkt(arr_subsref_arr,c[idx]=(b[idx] == 0) ? 0 : a[idx];)
// CUDA_BinaryFkt(carr_subsref_arr,c[idc]=(b[idx] == 0) ? 0 : a[idc]; c[idc+1]=(b[idx] == 0) ? 0 : a[idc+1];)
// CUDA_BinaryFkt(arr_subsasgn_arr,if (b[idx] == 0) c[idx] = a[idx];)
// CUDA_BinaryFkt(carr_subsasgn_arr,if (b[idx] == 0) {c[idc] = a[idc];c[idc+1] = a[idc+1];})
CUDA_MaskIdx(arr_subsref_arr,c[mask_idx]=a[idx];)
CUDA_MaskIdx(carr_subsref_arr,c[2*mask_idx]=a[2*idx]; c[2*mask_idx+1]=a[2*idx+1];)
CUDA_MaskIdx(arr_subsasgn_arr,a[idx]=c[mask_idx];)
CUDA_MaskIdx(carr_subsasgn_arr,a[2*idx]=c[2*mask_idx]; a[2*idx+1]=c[2*mask_idx+1];)

// diagonal matrix generation
CUDA_3DFkt(arr_diag_set,  size_t iddt=ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1]); c[iddt]=a[ids];)
CUDA_3DFkt(carr_diag_set,  size_t idcdt=2*(ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1])); c[idcdt]=a[2*ids];c[idcdt+1]=a[2*ids*1];)
CUDA_3DFkt(arr_diag_get,  size_t iddt=ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1]); a[ids]=c[iddt];)
CUDA_3DFkt(carr_diag_get,  size_t idcdt=2*(ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1])); a[2*ids]=c[idcdt];a[2*ids*1]=c[idcdt+1];)

// referencing and assignment with index vectors.No Index checking performed
// The code below "misuses" the CUDA_BinaryFkt macro to subreference or sub-assign from index lists
CUDA_BinaryFkt(arr_subsref_vec,{c[idx]=a[(size_t) b[idx]];})
CUDA_BinaryFkt(carr_subsref_vec,{c[2*idx]=a[2*((size_t) b[idx])];c[2*idx+1]=a[2*((size_t) b[idx])+1];})

CUDA_BinaryFkt(arr_subsasg_vec,{c[(size_t) b[idx]]=a[idx];})
CUDA_BinaryFkt(carr_subsasg_vec,{c[2*((size_t) b[idx])]=a[2*idx];c[2*((size_t) b[idx])+1]=a[2*idx+1];})

// one-D index operations. Note that the order is changed to remain compatible in the allocation of array c
// The NAN are needed to generate NaNs for wrong accesses.
//CUDA_IndexFkt(arr_subsref_ind,{if ((idx<M)&&(idx>=0)) {size_t myind=(size_t) b[idx];((myind<N)&&(myind>=0))?(c[idx]=a[myind]):c[idx]=NAN;} else c[idx]=NAN;})
//CUDA_IndexFkt(carr_subsref_ind,{if ((idx<M)&&(idx>=0)) {size_t myindC=2*(size_t) b[idx];((myindC<2*N)&&(myindC>=0))?(c[2*idx]=a[myindC],c[2*idx+1]=a[myindC+1]):(c[2*idx]=NAN,c[2*idx+1]=NAN);} else {c[2*idx]=NAN;c[2*idx+1]=NAN;}})

// The function below accepts a 2D index matrix (b) where each row is a list of indices corresponding to this dimension. The size of this matrix should have been adapted to the longest index list.

//CUDA_IndexFktND(arr_subsrefND_ind,CoordsNDFromIdx(idx,sSize,pos);for(int _d=0;_d<CUDA_MAXDIM;_d++){if ((idx<M)) {size_t myind=(size_t) b[idx];((myind<N))?(c[idx]=a[myind]):c[idx]=NAN;} else c[idx]=NAN;})

// CUDA_UnaryFktIntVec(arr_circshift_vec,CoordsNDFromIdx(idx,sSize,pos);for(int _d=0;_d<CUDA_MAXDIM;_d++){pos.s[_d]-=b.s[_d];}long long ids=0;IdxNDFromCoords(pos,sSize,ids);c[idx]=a[ids];)  // a[idx]


CUDA_IndexFkt(arr_subsref_ind,{if ((idx<M)) {size_t myind=(size_t) b[idx];((myind<N))?(c[idx]=a[myind]):c[idx]=NAN;} else c[idx]=NAN;})
//CUDA_IndexFkt(carr_subsref_ind,{if ((idx<M)&&(idx>=0)) {size_t myindC=2*(size_t) b[idx];((myindC<2*N)&&(myindC>=0))?(c[2*idx]=a[myindC],c[2*idx+1]=a[myindC+1]):(c[2*idx]=NAN,c[2*idx+1]=NAN);} else {c[2*idx]=NAN;c[2*idx+1]=NAN;}})
CUDA_IndexFkt(carr_subsref_ind,{if ((idx<M)) {size_t myindC=2*(size_t) b[idx];((myindC<2*N))?(c[2*idx]=a[myindC],c[2*idx+1]=a[myindC+1]):(c[2*idx]=NAN,c[2*idx+1]=NAN);} else {c[2*idx]=NAN;c[2*idx+1]=NAN;}})

CUDA_IndexFkt(arr_subsasgn_ind,{size_t myind=(size_t) b[idx]; if ((idx<M)) {((myind<N))?(c[myind]=a[idx]):0;} else c[myind]=NAN;})
CUDA_IndexFkt(carr_subsasgn_ind,{size_t myindC=2*(size_t) b[idx]; if ((idx<M)) {((myindC<2*N))?(c[myindC]=a[2*idx],c[myindC+1]=a[2*idx+1]):0;} else {c[idx]=NAN;c[2*idx+1]=NAN;}})

CUDA_UnaryFktConst(arr_subsasgn_const,{((idx<N))?(c[(size_t) a[idx]]=b):0;})
CUDA_UnaryFktConstC(arr_subsasgn_Cconst,{((idx<N))?(c[(size_t) a[idx]]=br):0;})  // This should not happen. If it does only the real part is kept.
CUDA_UnaryFktConstC(carr_subsasgn_const,{((idx<N))?(c[2*((size_t) a[idx])]=br,c[2*((size_t) a[idx])+1])=bi:0;})

// binary logical operations

CUDA_BinaryFkt(arr_or_arr,{c[idx]=(float) (a[idxA]!=0) || (b[idxB]!=0);})
CUDA_UnaryFktConst(arr_or_const,{c[idx]=(float) (a[idx]!=0) || (b!=0);})
CUDA_UnaryFktConst(const_or_arr,{c[idx]=(float) (b!=0) || (a[idx]!=0);})

CUDA_BinaryFkt(arr_and_arr,{c[idx]=(float) (a[idxA]!=0) && (b[idxB]!=0);})
CUDA_UnaryFktConst(arr_and_const,{c[idx]=(float) (a[idx]!=0) && (b!=0);})
CUDA_UnaryFktConst(const_and_arr,{c[idx]=(float) (b!=0) && (a[idx]!=0);})

// Unary logical operations
CUDA_UnaryFkt(not_arr,c[idx]=(a[idx] == 0);)

// Unary sign operation
CUDA_UnaryFkt(sign_arr,c[idx]=sign(a[idx]);)  // (a[idx] > 0)?1 :((a[idx]<0)?-1:0);
CUDA_UnaryFkt(sign_carr,size_t idc=2*idx; float absc=sqrt(a[idc]*a[idc]+a[idc+1]*a[idc+1]); if (absc==0) {c[idc]=0;c[idc+1]=0;} else {c[idc]=a[idc]/absc;c[idc+1]=a[idc+1]/absc;})

// Comparison
CUDA_BinaryFkt(arr_smaller_arr,c[idx]=a[idxA]<b[idxB];)
CUDA_UnaryFktConst(arr_smaller_const,c[idx]=a[idx]<b;)
CUDA_UnaryFktConst(const_smaller_arr,c[idx]=b<a[idx];)

CUDA_BinaryFkt(arr_larger_arr,c[idx]=a[idxA]>b[idxB];)
CUDA_UnaryFktConst(arr_larger_const,c[idx]=a[idx]>b;)
CUDA_UnaryFktConst(const_larger_arr,c[idx]=b>a[idx];)

CUDA_BinaryFkt(arr_smallerequal_arr,c[idx]=a[idxA]<=b[idxB];)
CUDA_UnaryFktConst(arr_smallerequal_const,c[idx]=a[idx]<=b;)
CUDA_UnaryFktConst(const_smallerequal_arr,c[idx]=b<=a[idx];)

CUDA_BinaryFkt(arr_largerequal_arr,c[idx]=a[idxA]>=b[idxB];)
CUDA_UnaryFktConst(arr_largerequal_const,c[idx]=a[idx]>=b;)
CUDA_UnaryFktConst(const_largerequal_arr,c[idx]=b>=a[idx];)

// equals will always output a real valued array
CUDA_BinaryFkt(arr_equals_arr,c[idx]=(a[idxA]==b[idxB]);)
CUDA_BinaryFkt(carr_equals_carr, size_t idcA=2*idxA;size_t idcB=2*idxB; c[idx]=(a[idcA]==b[idcB]) && (a[idcA+1]==b[idcB+1]);)
CUDA_BinaryFkt(carr_equals_arr,size_t idcA=2*idxA; c[idx]=(a[idcA]==b[idxB]) && (a[idcA+1] == 0);)
CUDA_BinaryFkt(arr_equals_carr,size_t idcB=2*idxB; c[idx]=(a[idxA]==b[idcB]) && (b[idcB+1] == 0);)
CUDA_UnaryFktConst(arr_equals_const,c[idx]=(a[idx]==b);)
CUDA_UnaryFktConst(const_equals_arr,c[idx]=(b==a[idx]);)
CUDA_UnaryFktConstC(carr_equals_const,size_t idc=2*idx; c[idx]=(a[idc]==br) && (a[idc+1]==bi);)
CUDA_UnaryFktConstC(const_equals_carr,size_t idc=2*idx; c[idx]=(br==a[idc]) && (bi==a[idc+1]);)
CUDA_UnaryFktConstC(arr_equals_Cconst,c[idx]=(a[idx]==br) && (bi==0);)
CUDA_UnaryFktConstC(Cconst_equals_arr,c[idx]=(br==a[idx]) && (bi==0);)

// not equals will always output a real valued array
CUDA_BinaryFkt(arr_unequals_arr,c[idx]=(a[idxA]!=b[idxB]);)
CUDA_BinaryFkt(carr_unequals_carr, size_t idcA=2*idxA;size_t idcB=2*idxB; c[idx]=(a[idcA]!=b[idcB]) || (a[idcA+1]!=b[idcB+1]);)
CUDA_BinaryFkt(carr_unequals_arr, size_t idcA=2*idxA;c[idx]=(a[idcA]!=b[idxB]) || (a[idcA+1] != 0);)
CUDA_BinaryFkt(arr_unequals_carr,size_t idcB=2*idxB;c[idx]=(a[idxA]!=b[idcB]) || (b[idcB+1] != 0);)
CUDA_UnaryFktConst(arr_unequals_const,c[idx]=(a[idx]!=b);)
CUDA_UnaryFktConst(const_unequals_arr,c[idx]=(b!=a[idx]);)
CUDA_UnaryFktConstC(carr_unequals_const,c[idx]=(a[2*idx]!=br) || (a[2*idx+1]!=bi);)
CUDA_UnaryFktConstC(const_unequals_carr,c[idx]=(br!=a[2*idx]) || (bi!=a[2*idx+1]);)
CUDA_UnaryFktConstC(arr_unequals_Cconst,c[idx]=(a[idx]!=br) || (bi!=0);)
CUDA_UnaryFktConstC(Cconst_unequals_arr,c[idx]=(br!=a[idx]) || (bi!=0);)

// other Unary oparations
CUDA_UnaryFkt(uminus_arr,c[idx]=-a[idx];)
CUDA_UnaryFkt(uminus_carr,size_t idc=2*idx; c[idc]=-a[idc];c[idc+1]=-a[idc+1];)   // negates real and imaginary part

CUDA_UnaryFkt(round_arr,c[idx]=round(a[idx]);)
CUDA_UnaryFkt(round_carr,size_t idc=2*idx; c[idc]=round(a[idc]);c[idc+1]=round(a[idc+1]);)   // negates real and imaginary part

CUDA_UnaryFkt(floor_arr,c[idx]=floor(a[idx]);)
CUDA_UnaryFkt(floor_carr,size_t idc=2*idx; c[idc]=floor(a[idc]);c[idc+1]=floor(a[idc+1]);)   // negates real and imaginary part

CUDA_UnaryFkt(ceil_arr,c[idx]=ceil(a[idx]);)
CUDA_UnaryFkt(ceil_carr,size_t idc=2*idx; c[idc]=ceil(a[idc]);c[idc+1]=ceil(a[idc+1]);)   // negates real and imaginary part

CUDA_UnaryFkt(exp_arr,c[idx]= exp(a[idx]);)
CUDA_UnaryFkt(exp_carr,size_t idc=2*idx; float len=exp(a[idc]);c[idc]=len*cos(a[idc+1]);c[idc+1]=len*sin(a[idc+1]);)

CUDA_UnaryFkt(sin_arr,c[idx]= sin(a[idx]);)
CUDA_UnaryFkt(sin_carr,size_t idc=2*idx; c[idc]=sin(a[idc])*cosh(a[idc+1]);c[idc+1]=cos(a[idc])*sinh(a[idc+1]);)

CUDA_UnaryFkt(cos_arr,c[idx]= cos(a[idx]);)
CUDA_UnaryFkt(cos_carr,size_t idc=2*idx; c[idc]=cos(a[idc])*cosh(a[idc+1]);c[idc+1]=sin(a[idc])*sinh(a[idc+1]);)

CUDA_UnaryFkt(tan_arr,c[idx]= tan(a[idx]);)

CUDA_UnaryFkt(sinh_arr,c[idx]= sinh(a[idx]);)
CUDA_UnaryFkt(sinh_carr,size_t idc=2*idx; c[idc]=sinh(a[idc])*cos(a[idc+1]);c[idc+1]=cosh(a[idc])*sin(a[idc+1]);)

CUDA_UnaryFkt(cosh_arr,c[idx]= cosh(a[idx]);)
CUDA_UnaryFkt(cosh_carr,size_t idc=2*idx; c[idc]=cosh(a[idc])*cos(a[idc+1]);c[idc+1]=sinh(a[idc])*sin(a[idc+1]);)

CUDA_UnaryFkt(sinc_arr, c[idx]= (a[idx] != 0) ? sin(a[idx])/a[idx] : 1.0;)
CUDA_UnaryFkt(sinc_carr,size_t idc=2*idx; c[idc]=0;c[idc+1]=0;) 
// c[idc]= (a[idc] == 0) ? sin(a[idc])*cosh(a[idc+1])/a[idc] : cosh(a[idc+1]);c[idc+1]= (a[idc] == 0) ? cos(a[idc])*sinh(a[idc+1])/a[idc] : sinh(a[idc+1]);)

// besselj, but order will be integer only:
CUDA_BinaryFkt(arr_besselj_arr,{c[idx]=jnf(size_t(a[idxA]),b[idxB]);})
CUDA_UnaryFktConst(arr_besselj_const,{c[idx]=jnf(size_t(a[idx]),b);})
CUDA_UnaryFktConst(const_besselj_arr,{c[idx]=jnf(size_t(b),a[idx]);})

// atan2 only for real inputs
CUDA_BinaryFkt(arr_atan2_arr,{c[idx]=atan2(b[idx],a[idx]);})
CUDA_UnaryFktConst(arr_atan2_const,{c[idx]=atan2(a[idx],b);})
CUDA_UnaryFktConst(const_atan2_arr,{c[idx]=atan2(b,a[idx]);})

CUDA_UnaryFkt(log_arr,c[idx]=log(a[idx]);)
CUDA_UnaryFkt(log_carr,c[2*idx]=log(a[2*idx]);c[2*idx+1]=0;)   //  not implemented

CUDA_UnaryFkt(abs_arr,c[idx]= (a[idx] > 0) ? a[idx] : -a[idx];)
CUDA_UnaryFkt(abs_carr,size_t idc=2*idx; c[idx]=sqrt(a[idc]*a[idc]+a[idc+1]*a[idc+1]);)

CUDA_UnaryFkt(conj_arr,c[idx]=a[idx];)
CUDA_UnaryFkt(conj_carr,size_t idc=2*idx; c[idc]=a[idc];c[idc+1]=-a[idc+1];)  // only affects the imaginary part

CUDA_UnaryFkt(sqrt_arr,c[idx]= sqrt(a[idx]);)
// funny expression below is the sign function ((x>0)-(x<0))
CUDA_UnaryFkt(sqrt_carr,size_t idc=2*idx; float L=sqrt(a[idc]*a[idc]+a[idc+1]*a[idc+1]); c[idc]=sqrt((L+a[idc])/2);c[idc+1]=((a[idc+1]>0)-(a[idc+1])<0)*sqrt((L-a[idc])/2);)

// Unary functions resulting in just a single value
CUDA_UnaryFkt(isIllegal_arr,if (isnan(a[idx]) || isinf(a[idx]) ) c[0]=1;)
CUDA_UnaryFkt(isIllegal_carr,if (a[2*idx+1]!=0 || isnan(a[2*idx]) || isnan(a[2*idx+1]) || isinf(a[2*idx]) || isinf(a[2*idx+1]) ) c[0]=1;)

CUDA_UnaryFkt(any_arr,if (a[idx]!=0) c[0]=1;)
CUDA_UnaryFkt(any_carr,if (a[2*idx]!=0 || a[2*idx+1]!=0) c[0]=1;)

// Binary functions with real valued input returning always complex arrays
CUDA_BinaryFkt(arr_complex_arr,c[2*idx]=a[idxA];c[2*idx+1]=b[idxB];)
CUDA_UnaryFktConst(arr_complex_const,c[2*idx]=a[idx];c[2*idx+1]=b;)
CUDA_UnaryFktConst(const_complex_arr,c[2*idx]=b;c[2*idx+1]=a[idx];)

// unary functions returning always real valued arrays

CUDA_UnaryFkt(real_arr,c[idx]=a[idx];)
CUDA_UnaryFkt(real_carr,c[idx]=a[2*idx];)

CUDA_UnaryFkt(imag_arr,c[idx]=0;)
CUDA_UnaryFkt(imag_carr,c[idx]=a[2*idx+1];)

CUDA_UnaryFkt(phase_arr,c[idx]=0;)
CUDA_UnaryFkt(phase_carr,c[idx]=atan2(a[2*idx+1],a[2*idx]);)

CUDA_UnaryFkt(isnan_arr,c[idx]=(float) isnan(a[idx]);)
CUDA_UnaryFkt(isnan_carr,c[idx]=(float) (isnan(a[2*idx])||isnan(a[2*idx+1]));)   // is not a number

CUDA_UnaryFkt(isinf_arr,c[idx]=(float) isinf(a[idx]);)
CUDA_UnaryFkt(isinf_carr,c[idx]=(float) (isinf(a[2*idx])||isinf(a[2*idx+1]));)   // is infinite

CUDA_UnaryFktIntVec(arr_circshift_vec,CoordsNDFromIdx(idx,sSize,pos);for(int _d=0;_d<CUDA_MAXDIM;_d++){pos.s[_d]-=b.s[_d];}long long ids=0;IdxNDFromCoords(pos,sSize,ids);c[idx]=a[ids];)  // a[idx]
CUDA_UnaryFktIntVec(carr_circshift_vec,CoordsNDFromIdx(idx,sSize,pos);for(int _d=0;_d<CUDA_MAXDIM;_d++){pos.s[_d]-=b.s[_d];}long long ids=0;IdxNDFromCoords(pos,sSize,ids);c[2*idx]=a[2*ids];c[2*idx+1]=a[2*ids+1];)

// In code below, the loop runs over the source dimensions. The array sizes are still set to the source sizes and will be (again) adjusted later
CUDA_UnaryFktIntVec(arr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos);
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=0;}
        for(_d=0;_d<CUDA_MAXDIM;_d++){
                if (b.s[_d]<CUDA_MAXDIM) {
                        dSize.s[_d]=sSize.s[b.s[_d]]; posnew.s[_d] = pos.s[b.s[_d]];}
                } 
        size_t idd=0;IdxNDFromCoords(posnew,dSize,idd);c[idd]=a[idx];}) // a[idx]

CUDA_UnaryFktIntVec(carr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos);
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=0;}
        for(_d=0;_d<CUDA_MAXDIM;_d++){
                if (b.s[_d]<CUDA_MAXDIM)  {
                        dSize.s[_d]=sSize.s[b.s[_d]]; posnew.s[_d] = pos.s[b.s[_d]];}
                }
        size_t idd=0;IdxNDFromCoords(posnew,dSize,idd);c[2*idd]=a[2*idx];c[2*idd+1]=a[2*idx+1];}) 
/*
CUDA_UnaryFktIntVec(arr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos); \
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=pos.s[_d];} \
        for(_d=0;_d<CUDA_MAXDIM;_d++){ \
                if (b.s[_d]<CUDA_MAXDIM && b.s[_d]>=0) { \
                        dSize.s[b.s[_d]]=sSize.s[_d]; posnew.s[b.s[_d]] = pos.s[_d];} \
                } \
        size_t idd=0;IdxNDFromCoords(posnew,dSize,idd);c[idd]=a[idx];}) 

CUDA_UnaryFktIntVec(carr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos); \
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=pos.s[_d];} \
        for(_d=0;_d<CUDA_MAXDIM;_d++){ \
                if (b.s[_d]<CUDA_MAXDIM && b.s[_d]>=0) { \
                        dSize.s[b.s[_d]]=sSize.s[_d]; posnew.s[b.s[_d]] = pos.s[_d];} \
                } \
        size_t idd=0;IdxNDFromCoords(posnew,dSize,idd);c[2*idd]=a[2*idx];c[2*idd+1]=a[2*idx+1];}) 
*/
        
CUDA_Fkt2Vec(arr_xyz_2vec,CoordsNDFromIdx(idx,sSize,pos);float val=0;for(int _d=0;_d<CUDA_MAXDIM;_d++){val += vec1.s[_d]+pos.s[_d]*(vec2.s[_d]-vec1.s[_d])/sSize.s[_d];} c[idx]=val;)  // a[idx]
CUDA_Fkt2Vec(arr_rr_2vec,CoordsNDFromIdx(idx,sSize,pos);float val=0;for(int _d=0;_d<CUDA_MAXDIM;_d++){val += Sqr(vec1.s[_d]+pos.s[_d]*(vec2.s[_d]-vec1.s[_d])/sSize.s[_d]);} c[idx]=sqrt(val);)  // a[idx]
CUDA_Fkt2Vec(arr_phiphi_2vec,CoordsNDFromIdx(idx,sSize,pos); c[idx]=atan2(vec1.s[0]+pos.s[0]*(vec2.s[0]-vec1.s[0])/sSize.s[0],vec1.s[1]+pos.s[1]*(vec2.s[1]-vec1.s[1])/sSize.s[1]);)  // phiphi

// Now include all the user-defined functions
// #include "user/user_cu_code.inc"
#include "user_cu_code.inc"


__global__ void set_arr(float b, float * c, size_t N)                          
{                                                          
   size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); 
   if(idx>=N) return;   
   c[idx]=b;
}                                                                   
extern "C" const char * CUDAset_arr(float b, float * c, size_t N)  
{                                                                       
    hipError_t myerr;                                                
	size_t blockSize;dim3 nBlocks;                                         
    MemoryLayout(N,blockSize,nBlocks)                                    
	set_arr<<<nBlocks,blockSize>>>(b,c,N);                            
  myerr=hipGetLastError();                                             
  if (myerr != hipSuccess)                                             
      return hipGetErrorString(myerr);                                 
  return 0;                                                                   
}                                                                       

__global__ void set_carr(float br, float bi, float * c, size_t N)               
{                                                                   
   size_t idx=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); 
    if(idx>=N) return;   
    size_t idc=idx*2;                                                  
	c[idc]=br;c[idc+1]=bi;
}

extern "C" const char * CUDAset_carr(float br, float bi, float * c, size_t N)
{                                                                       
    hipError_t myerr;                                             
	size_t blockSize;dim3 nBlocks;                                         
    MemoryLayout(N,blockSize,nBlocks)                                    
	set_carr<<<nBlocks,blockSize>>>(br,bi,c,N);                        
  myerr=hipGetLastError();                                             
  if (myerr != hipSuccess)                                             
      return hipGetErrorString(myerr);                                 
  return 0;                                                             
}                                                                       

// function below is used to check whether CUIMAGE_REDUCE_THREADS is set correctly 
extern "C" int ReduceThreadsDef(void) {
    return CUIMAGE_REDUCE_THREADS;
}

extern "C" const char * SetDeviceProperties(void) {
    hipError_t myerr;                                             
    int dev=0;
    hipGetDevice(&dev);
    myerr=hipGetDeviceProperties(&prop,dev);
    if (myerr != hipSuccess)  
        return hipGetErrorString(myerr);
    return 0;
}


extern "C" size_t CUDAmaxSize(void) {
    int dev=0;
    hipGetDevice(&dev);
    struct hipDeviceProp_t prop;
    int status=hipGetDeviceProperties(&prop,dev);

    // return prop.maxThreadsPerBlock;  // 512
    // return prop.multiProcessorCount;   // 30
    // return prop.warpSize;   // 32
    // return prop.maxThreadsDim[0];   // 512  = max blocksize
    // return prop.maxGridSize[0];   // 65535  = max GridSize = max nBlocks?
    return ((size_t)prop.maxGridSize[0])*((size_t)prop.maxGridSize[1])*((size_t)prop.maxThreadsDim[0]);   // maximally 2D grids are currently allowed.
}


__global__ void
arr_times_const_checkerboard(float*a,float b, float * c, size_t N, size_t sx,size_t sy,size_t sz)
{
    size_t ids=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); // which source array element do I have to deal with?
	if(ids>=N) return;  // not in range ... quit

	size_t px=(ids/2)%sx;   // my x pos
	size_t py=(ids/2)/sx;   // my y pos
    float minus1=(1-2*((px+py)%2));
	c[ids]=a[ids]*b*minus1;
}

extern "C" int CUDAarr_times_const_checkerboard(float * a, float b, float * c, size_t * sizes, int dims)  // multiplies with a constand and scrambles the array
{
    size_t sx=sizes[0],sy=1,sz=1;
    if (dims>1)
        sy=sizes[1];
    if (dims>2)
        sz=sizes[2];
    size_t N=sx*sy*sz*2;  // every pair will be processed exactly once
	size_t blockSize;dim3 nBlocks;                                         
    MemoryLayout(N,blockSize,nBlocks)                                    
	arr_times_const_checkerboard<<<nBlocks,blockSize>>>(a,b,c,N,sx,sy,sz);
	return 0;
}


/// cyclicly rotates datastack cyclic into positive direction in all coordinates by (dx,dy,dz) voxels
/// simple version with all processors dealing with exactly one element
__global__ void
rotate2(float*a,float b, float * c, size_t sx,size_t sy,size_t sz, long long dx, long long dy, long long dz)
{
  size_t ids=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); // id of this processor
  long long x=(ids + dx)%sx;  // advance by the offset steps along the chain
  long long y=(ids/sx + dy)%sy;
  long long z=(ids/(sx*sy) + dz)%sz;
  size_t idd=x+sx*y+sx*sy*z;
  if(ids>=sx*sy*sz) return;
  // float tmp = a[ids];
  // __syncthreads();             // nice try but does not work !
  c[idd] = b*a[ids];
}

/// cyclicly rotates datastack cyclic into positive direction in all coordinates by (dx,dy,dz) voxels
__global__ void
rotate(float*a,float b, float * c, size_t sx,size_t sy,size_t sz, size_t dx, size_t dy, size_t dz, size_t ux, size_t uy, size_t uz)
{
  // id of this processor
  size_t id=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); 

  size_t Processes=blockDim.x * gridDim.x;
  size_t chains=ux*uy*uz; // total number of independent chains
  size_t N=sx*sy*sz;  // total size of array, has to be chains*length_of_chain
  size_t length=N/chains;  // chain length
  size_t steps=N/Processes;  // this is how many steps each processor has to do

  size_t step,nl,nx,ny,nz,x,y,z,i,idd;
  float swp, nswp;

//if (id != 0)   return;
//for (id=0;id<Processes;id++)
{
  step=steps*id;   // my starting step as the id times the number of steps
  nl=step%length;  // current position in chain length
  nx=(step/length)%ux;  // current position in unit cell x
  ny=(step/(length*ux))%uy;  // current position in unit cell y
  nz=(step/(length*ux*uy))%uz;  // current position in unit cell z
  i=0;

  //if (step/steps != 4 && step/steps != 5) return;

  while(nz<uz)
   {
      while(ny<uy)
        {
          while (nx<ux)
            {
            x=(nx+nl*dx)%sx;  // advance by the offset steps along the chain
            y=(ny+nl*dy)%sy;
            z=(nz+nl*dz)%sz;
            idd=x+sx*y+sx*sy*z;
            if (i < steps) {
                swp=a[idd]; 
                // a[idd]=a[idd]+0.1;
                __syncthreads();
            }
            while (nl<length-1)
                {
                  if (i > steps-1)
                    goto nextProcessor; // return;
                  if (step >= N)  // this thread has reached the end of the total data to process
                    goto nextProcessor; // return;
                  step++;
                  x = (x+dx)%sx; // new position
                  y = (y+dy)%sy;
                  z = (z+dz)%sz;
                  idd=x+sx*y+sx*sy*z;
                  if (i < steps-1) {
                    nswp=a[idd];
                    __syncthreads();
                    //a[idd]=a[idd]+0.1;
                    }

                  c[idd]=swp+0.1; // c[idd]+ny+0.1; // c[idd]+i; // swp+0.1; // c[idd]+(step/steps);
                  i++; // counts number of writes
                  if (i > steps-1)
                    goto nextProcessor; // return;
                  nl++;
                  if (i < steps) {
                  swp=nswp;
                  }
                }
            nx++; nl=0;
            //if (nx < ux) {
            x = (x+dx)%sx; // new position
            y = (y+dy)%sy;
            z = (z+dz)%sz;
            idd=x+sx*y+sx*sy*z;
            c[idd]=swp+0.1; // no need to save this value as this is the end of the line
            //}
            i++; 
            if (i > steps-1)
                goto nextProcessor; // return;
            // if (nx <ux) x=(x+1)%sx;
            }
        ny++;
        // if (ny <uy) y=(y+1)%sy;
        nx=0;x=0;
        }
    nz++;
    // if (nz <uz) z=(z+1)%sz;
    ny=0;y=0;
    }
nextProcessor:
nz=0;
}
return;
}

size_t gcd(size_t a, size_t b) // greatest commod divisor by recursion
{ 
   return ( b == 0 ? a : gcd(b, a % b) ); 
}

extern "C" int CUDAarr_times_const_rotate(float * a, float b, float * c, size_t * sizes, int dims, int complex,int direction)  // multiplies with a constand and cyclilcally rotates the array using the chain algorithm
{
    // printf("TestING\n");   % Does NOT work!
    long long sx=1,sy=1,sz=1;
    if (dims>0)
        sx=sizes[0];
    if (dims>1)
        {sx=sizes[0];sy=sizes[1];}
    if (dims>2)
        sz=sizes[2]; 

    long long dx=(sx+direction*sx/2)%sx,dy=(sy+direction*sy/2)%sy,dz=(sz+direction*sz/2)%sz;  // how much to cyclically rotate
    if (complex) {sx=sx*2;dx=dx*2;}
    //printf("sx %d sy %d dx %d dy %d\n",sx,sy,dx,dy);

    // calculate the length of each swapping chain
    long long ux=gcd(sx,dx);  // unit cell in x. Any repeat along y directions will be also a repeat in x. Chain length is sx/ux
    // size_t lx=sx/ux; // how many accesses to get one round in x
    long long uy=gcd(((sx/ux)*dy%sy),sy); // how many times must the first chain be repeated to form a longer chain. This defines unit cell y
    long long uz=gcd(((sy/uy)*dz%sz),sz); // similar for z
    long long length=sx*sy*sz/(ux*uy*uz);  // chain length

    // in one dimension the gcd=greatest common divisor, would mean that one has to start task at position 0 ... gcd-1
    // in several dimensions even completing one round leaving a spacing at gcd does not mean that this is a complete loop
    // however it could be a complete loop. The number of steps that where performed in the lower dimension are s/gcd before reaching the beginning again
    // with the size of the dimension s. If we are at the same startingpoint in the next dimension the chain is complete.
    // So the number of times a super chain (in 2D) must be executed is sy/gcd(sy,s/gcd(sx,dx))
    int dev=0;
    hipGetDevice(&dev);
    struct hipDeviceProp_t prop;
    int status=hipGetDeviceProperties(&prop,dev);

    long long m=1;
    if (ux>uy)
        m=ux;
    else
        m=uy;
    if (uz>m)
        m=uz;
    if (length>m)
        m=length;

    //size_t blockSize=1; // prop.warpSize; // ux*uy*uz;
    //size_t nBlocks=m;	// add extra block if N can't be divided by blockSize
    
    //    rotate<<<nBlocks,blockSize>>>(a,b,c,sx,sy,sz,dx,dy,dz,ux,uy,uz);  // get unit cell sizes

    size_t N=sx*sy*sz; // includes the space for complex numbers
	size_t blockSize;dim3 nBlocks;                                         
                                                                //    printf("BlockSize %d, ux %d, uy %d, uz %d\n",blockSize,ux,uy,uz);
    // unfortunately we have to do it out of place.
    MemoryLayout(N,blockSize,nBlocks)                                    
    // printf("rotate 2 call: (%zd %zd %zd %lld %lld %lld)\n",sx,sy,sz,dx,dy,dz);
    if (a == c)
    {
        float * d =0;
        int status=hipMalloc((void **) &d, N*sizeof(float));
        hipMemcpy(d,a, N*sizeof(float),hipMemcpyDeviceToDevice);
        rotate2 <<<nBlocks,blockSize>>>(d,b,c,sx,sy,sz,dx,dy,dz);
        hipFree(d);
    }
    else
        rotate2 <<<nBlocks,blockSize>>>(a,b,c,sx,sy,sz,dx,dy,dz);  // get unit cell sizes

	return prop.maxThreadsPerBlock;
}



__global__ void
arr_times_const_scramble(float*a,float b, float * c, size_t sx,size_t sy,size_t sz, size_t ox, size_t oy, size_t oz)
{
	// which source array element do I have to deal with?
    size_t pnum=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); 

	size_t px=pnum%(sx/2);   // my x pos of a complex number in the subarray
	size_t py=pnum/(sx/2);   // my y pos of a complex number
	if(px>=(sx/2) || py >= (sy/2)) return;  // not in range ... quit
    size_t ids=2*(px+py*sx);  /// offset to array start in floats
    size_t idd=2*((ox+px)+(oy+py)*sx);

    // echange two values using a tmp
    float tmpR = c[idd];
    float tmpI = c[idd+1];
    c[idd]=a[ids]; // (float)(ox+px); // 
    c[idd+1]=a[ids+1]; // (float)(oy+py); // 
    a[ids]=tmpR;
    a[ids+1]=tmpI;
}

__global__ void
array_copy(float*a, float * c, size_t mx, size_t my, size_t mz, size_t sx,size_t sy,size_t sz, size_t ox, size_t oy, size_t oz)  // copies between two memories with different strides
{
    size_t pnum=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x); 

	size_t px=pnum%(sx/2);   // my x pos of a complex number in the subarray
	size_t py=pnum/(sx/2);   // my y pos of a complex number
	if(px>=sx || py >= (sy/2)) return;  // not in range ... quit
    size_t ids=2*(px+py*sx);  /// offset to array start in floats
    size_t idd=2*((ox+px)+(oy+py)*sx);

    // echange two values using a tmp
    float tmpR = c[idd];
    float tmpI = c[idd+1];
    c[idd]=a[ids]; // (float)(ox+px); // 
    c[idd+1]=a[ids+1]; // (float)(oy+py); // 
    a[ids]=tmpR;
    a[ids+1]=tmpI;
}


extern "C" int CUDAarr_times_const_scramble(float * a, float b, float * c, size_t * sizes, int dims)  // multiplies with a constand and scrambles the array
{
    size_t sx=sizes[0],sy=1,sz=1, iseven=1;
	size_t blockSize;dim3 nBlocks;                                         
    if (sx%2 == 1) iseven=0;
    if (dims>1) {
        sy=sizes[1];
        if (sy%2 == 1) iseven=0;
        }

    if (dims>2) {
        sz=sizes[2];
        if (sz%2 == 1) iseven=0;
        }
    size_t N=sx*sy*sz*2;  // every pair will be processed exactly once
    MemoryLayout(N,blockSize,nBlocks)                                    

    if (! iseven)
        {
            float * tmp=0;
            hipMalloc((void **) &tmp,sizeof(tmp[0])*(1+sx/2)*(1+sy/2));
        }
	arr_times_const_scramble<<<nBlocks,blockSize>>>(a,b,c,sx,sy,sz,sx/2,sy/2,0);
	arr_times_const_scramble<<<nBlocks,blockSize>>>(a+2*(sx/2),b,c+2*(sx/2),sx,sy,sz,-sx/2,sy/2,0);
	return 0;
}


// Here is some code for calculating the singular value decomposition of the trailing dimension in an array
// This code is adopted from matLib3D.h by stamatis.lefkimmiatis@epfl.ch
// and svd3D_decomp.cpp by emmanuel.soubies@epfl.ch

/***************************************************************************
  Let X be a NxMxKx6 matrix such that:
  
  P_mn = [X(n,m,k,1) X(n,m,k,2) X(n,m,k,3)
          X(n,m,k,2) X(n,m,k,4) X(n,m,k,5)
          X(n,m,k,3) X(n,m,k,5) X(n,m,k,6)] 
          
  is a symmetric matrix. Then the present function computes the eigenvalues
  E(n,m,k,1) E(n,m,k,2) E(n,m,k,3) and the eigenvector 
          V1 = [V(n,m,k,1) V(n,m,k,2)  V(n,m,k,3)] 
          V2 = [V(n,m,k,4) V(n,m,k,5)  V(n,m,k,6)] 
          V2 = [V(n,m,k,5) V(n,m,k,8)  V(n,m,k,9)]  
  Hence the function outputs two matrices E of size NxMxKx3 and V of size NxMxKx9.
  
  Copyright (C) 2017 E. Soubies emmanuel.soubies@epfl.ch

****************************************************************************/

#include "matlib3D.h"

__global__ void core_svd3D(float *X, float *Ye, float * Yv, size_t N){   // N is NOT the total size, but only the size excluding the last dimension (of size 3)
  size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);
  if(idd>=N) return;
  int k;
    double E[3];
	double D[3];   
    double V[9];
  
        for (k=0;k<3;k++)   // get the matrix value [X(1,1) X(2,1)=X(1,2), X(2,2)]
            V[k]=X[idd+N*k];
        for (k = 4; k < 6; k++)
            V[k] = X[idd+N*(k-1)];
        for (k = 7; k < 9; k++)
            V[k] = X[idd+N*(k-3)];
        V[3]=X[idd+N];
        V[6]=X[idd+N*2];
        
        tred2(V, D, E);
        tql2(V, D, E);
        
  		for (k=0;k<3;k++)  // set result
        	Ye[idd+N*k]=D[k];
        
        for (k=0;k<9;k++){
            Yv[idd+N*k]=V[k];
        }
}

extern "C" const char * CUDAsvd_last(float *X, float *Ye, float * Yv, size_t N)  // N is NOT the total size, but only the size excluding the last dimension (of size 3)
{
    hipError_t myerr;
	dim3 nBlocks;
    size_t blockSize=prop.maxThreadsPerBlock / 2; // To account for the many registers needed
    size_t numb=NBLOCKS(N,blockSize);
    if (numb<prop.maxGridSize[0])
        nBlocks.x=numb;
    else
        {nBlocks.x=(size_t)(sqrt((float)numb)+1);
    nBlocks.y=(size_t)(sqrt((float)numb)+1);}

	core_svd3D<<<nBlocks,blockSize>>>(X,Ye,Yv,N);
    myerr=hipGetLastError();
    if (myerr != hipSuccess)
        return hipGetErrorString(myerr);
    return 0;
}


__global__ void core_svd3D_recomp(float *Y, float *E, float * V, size_t N){   // N is NOT the total size, but only the size excluding the last dimension (of size 3)
  size_t idd=((blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x);
  if(idd>=N) return;
  int k;
  double ee[3];
	double vv[9];
	double tmp[6];  
  
    	for (k=0;k<3;k++)   // get the eigenvalues 
        	ee[k]=E[idd+N*k];
        for (k=0;k<9;k++)   // get the eigenvectors
        	vv[k]=V[idd+N*k];

        eigen3x3SymRec(tmp,vv,ee);
  		for (k=0;k<6;k++){  // set result
        	Y[idd+N*k]=tmp[k];
  		}
}

extern "C" const char * CUDAsvd3D_recomp(float *Y, float *E, float * V, size_t N)  // N is NOT the total size, but only the size excluding the last dimension (of size 3)
{
    hipError_t myerr;
	dim3 nBlocks;
    size_t blockSize=prop.maxThreadsPerBlock / 2; // To account for the many registers needed
    size_t numb=NBLOCKS(N,blockSize);
    if (numb<prop.maxGridSize[0])
        nBlocks.x=numb;
    else
        {nBlocks.x=(size_t)(sqrt((float)numb)+1);
    nBlocks.y=(size_t)(sqrt((float)numb)+1);}

	core_svd3D_recomp<<<nBlocks,blockSize>>>(Y,E,V,N);
    myerr=hipGetLastError();
    if (myerr != hipSuccess)
        return hipGetErrorString(myerr);
    return 0;
}

