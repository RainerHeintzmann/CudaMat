#include "hip/hip_runtime.h"
/************************* CudaMat ******************************************
 *   Copyright (C) 2008-2009 by Rainer Heintzmann                          *
 *   heintzmann@gmail.com                                                  *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; Version 2 of the License.               *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************
 * Compile with:
 * Windows:
system('"c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin\vcvars32.bat"')
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 9.0\VC\bin')

Window 64 bit:
system('nvcc -c cudaArith.cu -ccbin "c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I"c:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" ')

Linux:
 * File sudo vi /usr/local/cuda/bin/nvcc.profile
 * needs the flag -fPIC  in the include line
system('nvcc -c cudaArith.cu -v -I/usr/local/cuda/include/')
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "hipfft/hipfft.h"
#include "cudaArith.h"
#define IMUL(a, b) __mul24(a, b)

#define BLOCKSIZE 1024
//#define BLOCKSIZE 512
#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

// the real part is named ".x" and the imaginary ".y" in the hipfftComplex datatype
__device__ hipfftComplex cuda_resultVal;   // here real and complex valued results can be stored to be then transported to the host
__device__ int cuda_resultInt;   // here real and complex valued results can be stored to be then transported to the host
static float * TmpRedArray=0;   // This temporary array will be constructed on the device, whenever the first reduce operation is performed
static float * accum = 0;       // This is the corresponding array on the host side
static int CurrentRedSize=0;    // Keeps track of how much reduce memory is allocated on the device
static const int MinRedBlockSize=65536;    // defines the chunks of memory (in floats) which will be used

// below is blocksize for temporary array for reduce operations. Has to be a power of 2 in size
#define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 512
// #define CUIMAGE_REDUCE_THREADS 128
//#define CUIMAGE_REDUCE_BLOCKS  64

#define mysum(a,b) ((a)+(b))
#define maxCond(a,b) (((b)>(a)))
#define minCond(a,b) (((b)<(a)))

#define Sqr(a) ((a)*(a))

// below are code snippets used in other macros 
#define Coords3DFromIdx(idx,sSize)                                      \
  int x=(idx)%sSize.s[0];                                               \
  int y=(idx/sSize.s[0])%sSize.s[1];                                    \
  int z=(idx/(sSize.s[0]*sSize.s[1]))%sSize.s[2];                       

#define IdxFromCoords3D(x,y,z,dSize,dOffs) \
  unsigned int idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]);  \
  
#define CoordsNDFromIdx(idx,sSize,pos)                                  \
   SizeND pos;                                               \
   { unsigned int resid=idx;                                               \
  for(int _d=0;_d<CUDA_MAXDIM;_d++)                                     \
      if (resid > 0)                                                    \
        { pos.s[_d]=resid%sSize.s[_d];                                    \
          resid/=sSize.s[_d]; }                                         \
      else                                                              \
          pos.s[_d]=0;                                                    \
  }

#define IdxNDFromCoords(pos,dSize,idd)                                   \
  (idd)=0;                                                              \
  {                                                                     \
  unsigned int _Stride=1;                                                \
  for(int _d=0;_d<CUDA_MAXDIM;_d++)                                      \
  if (dSize.s[_d]>0) {                                                   \
          if (pos.s[_d] < 0)                                              \
              {(idd) += (dSize.s[_d]-((-pos.s[_d]) % dSize.s[_d])) *_Stride;}          \
          else                                                          \
              {(idd) += (pos.s[_d] % dSize.s[_d]) *_Stride;}  \
        _Stride *= dSize.s[_d]; }                                        \
}
   

// The partial reduction funciton below projects the data along one dimension
// the processors are assigned to the result image pixels
// CAVE: These versions can be slow, if the resulting data has is smaller than the number of processors
#define CUDA_PartRedMask(FktName, OP)               \
__global__ void FktName (float *in, float *out, float * mask, int N, int dSizeX, int sStrideX, int sStrideY, int ProjStride, int ProjSize){      \
  int idd=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  if(idd>=N) return;                                                    \
  int p;                                                                \
  int ids=(idd%dSizeX)*sStrideX+(idd/dSizeX)*sStrideY;                  \
  float accu=0.0;                                                       \
  int laterPix=0;                                                       \
  for (p=0;p<ProjSize;p++)                                              \
    {                                                                   \
      if (mask == 0 || mask[ids] != 0.0)                                \
        if (! laterPix)  {                                              \
            accu=in[ids];                                               \
            laterPix=1;                                                 \
        } else {                                                        \
            accu=OP(accu,in[ids]);                                      \
        }                                                               \
      ids += ProjStride;                                                \
    }                                                                   \
 out[idd] = accu;                                                       \
}                                                                       \
\
extern "C" const char * CUDA ## FktName(float *a, float * mask, float * c, int sSize[CUDA_MAXPROJ], int ProjDir)\
{                                                                       \
    hipError_t myerr;                                                  \
    int dSize[CUDA_MAXPROJ],d,N=1;                                      \
    for (d=0;d<CUDA_MAXPROJ;d++)  {dSize[d]=sSize[d]; }                 \
    dSize[ProjDir-1]=1;                                                 \
    for (d=0;d<CUDA_MAXPROJ;d++)  {N*=dSize[d];}                         \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    int ProjStride=0,ProjSize=0,sStrideX=0,sStrideY=0,dSizeX=0;         \
    if (ProjDir==1)                                                     \
        {ProjStride=1;ProjSize=sSize[0];dSizeX=sSize[1];sStrideX=sSize[0];sStrideY=sSize[0]*sSize[1];}\
    else if (ProjDir == 2)                                              \
        {ProjStride=sSize[0];ProjSize=sSize[1];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0]*sSize[1];}\
    else if (ProjDir == 3)                                              \
        {ProjStride=sSize[0]*sSize[1];ProjSize=sSize[2];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else if (ProjDir == 4)                                              \
        {ProjStride=sSize[0]*sSize[1]*sSize[2];ProjSize=sSize[3];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else if (ProjDir == 5)                                              \
        {ProjStride=sSize[0]*sSize[1]*sSize[2]*sSize[3];ProjSize=sSize[4];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else                                                                \
        return "Error: Unsupported projection direction";               \
	FktName<<<nBlocks,blockSize>>>(a,c,mask,N,dSizeX,sStrideX,sStrideY,ProjStride,ProjSize);\
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}

//   This is the same as the above but suited for complex numbers
#define CUDA_PartRedMaskCpx(FktName, OP)               \
__global__ void FktName (float *in, float *out, float * mask, int N, int dSizeX, int sStrideX, int sStrideY, int ProjStride, int ProjSize){      \
  int idd=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  if(idd>=N) return;                                                    \
  int p;                                                                \
  int ids=(idd%dSizeX)*sStrideX+(idd/dSizeX)*sStrideY;                  \
  float accu=0.0;                                                       \
  float accuI=0.0;                                                      \
  int laterPix=0;                                                       \
  for (p=0;p<ProjSize;p++)                                              \
    {                                                                   \
      if (mask == 0 || mask[ids] != 0.0)                                \
        if (! laterPix)  {                                              \
            accu=in[2*ids];                                             \
            accuI=in[2*ids+1];                                          \
            laterPix=1;                                                 \
        } else {                                                        \
            accu=OP(accu,in[2*ids]);                                      \
            accuI=OP(accuI,in[2*ids+1]);                                    \
        }                                                               \
      ids += ProjStride;                                                \
    }                                                                   \
 out[2*idd] = accu;                                                       \
 out[2*idd+1] = accuI;                                                       \
}                                                                       \
\
extern "C" const char * CUDA ## FktName(float *a, float * mask, float * c, int sSize[3], int ProjDir)\
{                                                                       \
     hipError_t myerr;                                                \
    int dSize[CUDA_MAXPROJ],d,N=1;                                      \
    for (d=0;d<CUDA_MAXPROJ;d++)  {dSize[d]=sSize[d]; }                 \
    dSize[ProjDir-1]=1;                                                 \
    for (d=0;d<CUDA_MAXPROJ;d++)  {N*=dSize[d];}                         \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    int ProjStride=0,ProjSize=0,sStrideX=0,sStrideY=0,dSizeX=0;         \
    if (ProjDir==1)                                                     \
        {ProjStride=1;ProjSize=sSize[0];dSizeX=sSize[1];sStrideX=sSize[0];sStrideY=sSize[0]*sSize[1];}\
    else if (ProjDir == 2)                                              \
        {ProjStride=sSize[0];ProjSize=sSize[1];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0]*sSize[1];}\
    else if (ProjDir == 3)                                              \
        {ProjStride=sSize[0]*sSize[1];ProjSize=sSize[2];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else if (ProjDir == 4)                                              \
        {ProjStride=sSize[0]*sSize[1]*sSize[2];ProjSize=sSize[3];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else if (ProjDir == 5)                                              \
        {ProjStride=sSize[0]*sSize[1]*sSize[2]*sSize[3];ProjSize=sSize[4];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else                                                                \
        return "Error: Unsupported projection direction";                     \
	FktName<<<nBlocks,blockSize>>>(a,c,mask,N,dSizeX,sStrideX,sStrideY,ProjStride,ProjSize);\
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}

// This partial reduction code keeps track of the index

#define CUDA_PartRedMaskIdx(FktName, OP)               \
__global__ void FktName (float *in, float *out, float * outIdx, float * mask, int N, int dSizeX, int sStrideX, int sStrideY, int ProjStride, int ProjSize){      \
  int idd=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  if(idd>=N) return;                                                    \
  int p;                                                                \
  int ids=(idd%dSizeX)*sStrideX+(idd/dSizeX)*sStrideY;                  \
  float accu=0.0;                                                       \
  float accuIdx=-1;                                                     \
  int laterPix=0;                                                       \
  for (p=0;p<ProjSize;p++)                                              \
    {                                                                   \
      if (mask == 0 || mask[ids] != 0.0)                                \
        if (! laterPix)  {                                              \
            accu=in[ids];                                               \
            accuIdx=p;                                                  \
            laterPix=1;                                                 \
        } else {                                                        \
            if (OP(accu,in[ids])) {accu=in[ids];accuIdx=p;}             \
        }                                                               \
      ids += ProjStride;                                                \
    }                                                                   \
 out[idd] = accu;                                                       \
 if (outIdx != 0)                                                       \
    outIdx[idd] = accuIdx;                                                 \
}                                                                       \
\
extern "C" const char * CUDA ## FktName(float *a, float * mask, float * c, float * cIdx, int sSize[5], int ProjDir)\
{                                                                       \
    hipError_t myerr;                                                  \
    int dSize[CUDA_MAXPROJ],d,N=1;                                      \
    for (d=0;d<CUDA_MAXPROJ;d++)  {dSize[d]=sSize[d]; }                 \
    dSize[ProjDir-1]=1;                                                 \
    for (d=0;d<CUDA_MAXPROJ;d++)  {N*=dSize[d];}                         \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    int ProjStride=0,ProjSize=0,sStrideX=0,sStrideY=0,dSizeX=0;         \
    if (ProjDir==1)                                                     \
        {ProjStride=1;ProjSize=sSize[0];dSizeX=sSize[1];sStrideX=sSize[0];sStrideY=sSize[0]*sSize[1];}\
    else if (ProjDir == 2)                                              \
        {ProjStride=sSize[0];ProjSize=sSize[1];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0]*sSize[1];}\
    else if (ProjDir == 3)                                              \
        {ProjStride=sSize[0]*sSize[1];ProjSize=sSize[2];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else if (ProjDir == 4)                                              \
        {ProjStride=sSize[0]*sSize[1]*sSize[2];ProjSize=sSize[3];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else if (ProjDir == 5)                                              \
        {ProjStride=sSize[0]*sSize[1]*sSize[2]*sSize[3];ProjSize=sSize[4];dSizeX=sSize[0];sStrideX=1;sStrideY=sSize[0];}\
    else                                                                \
        return "Error: Unsupported projection direction";               \
	FktName<<<nBlocks,blockSize>>>(a,c,cIdx,mask,N,dSizeX,sStrideX,sStrideY,ProjStride,ProjSize);\
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}

// Below is some reduction code adapted from the tips and tricks tutorial 
// https://www.sharcnet.ca/help/index.php/CUDA_tips_and_tricks
// FUNCTION BELOW IS SLOW AND DOES NOT WORK PROPERLY YET
#define CUDA_FullRedBin(FktName, OP)                                    \
__global__ void FktName (float *in, int N){                             \
  const int stride = CUIMAGE_REDUCE_THREADS;                    \
  const int start  = threadIdx.x;\
  __shared__ float accum[CUIMAGE_REDUCE_THREADS];               \
  int nTotalThreads=CUIMAGE_REDUCE_THREADS;                     \
  int thread2;                                                  \
                                                                \
  if (start >= CUIMAGE_REDUCE_THREADS) return;                   \
  if (start >= N) {accum[start]=0;return;}                      \
                                                                \
  accum[threadIdx.x] = in[start];                               \
  for (int ii=start+stride; ii < N; ii += CUIMAGE_REDUCE_THREADS)  { \
    accum[threadIdx.x] = OP(accum[threadIdx.x], in[ii]);        \
  }                                                             \
  __syncthreads();                                              \
                                                                \
/* Now entering the logaritmic reduction phase of the algorithm*/       \
while(nTotalThreads > 1)                                                \
{                                                                       \
  int halfPoint = (nTotalThreads >> 1);	/* divide by two */             \
  /* only the first half of the threads will be active. */              \
                                                                        \
  if (threadIdx.x < halfPoint)                                          \
  {  thread2 = threadIdx.x + halfPoint;                                   \
   /* Skipping the fictious threads blockDim.x ... blockDim_2-1 */      \
   if (thread2 < stride)                                            \
      accum[threadIdx.x]=OP(accum[threadIdx.x],accum[thread2]);         \
  }                                                                     \
  __syncthreads();                                                      \
  /* Reducing the binary tree size by two:  */                          \
  nTotalThreads = halfPoint;                                            \
}                                                                       \
  __syncthreads();                                              \
  if (threadIdx.x == 0)                                         \
          cuda_resultVal.x=accum[0];                            \
}                                                               \
extern "C" const char * CUDA ## FktName(float * a, int N, float * resp) \
{                                                               \
  int CUIMAGE_REDUCE_BLOCKS=NBLOCKS(N,CUIMAGE_REDUCE_THREADS);  \
  dim3 threadBlock(CUIMAGE_REDUCE_THREADS);                     \
  dim3 blockGrid(CUIMAGE_REDUCE_BLOCKS);                        \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, N);                    \
  if (hipGetLastError() != hipSuccess)                        \
      return hipGetErrorString(hipGetLastError());            \
                                                                \
  hipMemcpyFromSymbol(resp, HIP_SYMBOL(cuda_resultVal), sizeof(* resp));\
  if (hipGetLastError() != hipSuccess)                        \
      return hipGetErrorString(hipGetLastError());            \
  return 0;                                                     \
}


// Below is the reduction code of Wouter Caarls, modified
// This could potentially also be run sequentially over the remaining dimension

#define CUDA_FullRed(FktName, OP)                               \
__global__ void FktName (float *in, float *out, int N){      \
  const int stride = blockDim.x * gridDim.x;                    \
  const int start  = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;\
  __shared__ float accum[CUIMAGE_REDUCE_THREADS];               \
  if (start >= N) return;                                    \
                                                                \
  accum[threadIdx.x] = in[start];                               \
  for (int ii=start+stride; ii < N; ii += stride)  {         \
    accum[threadIdx.x] = OP(accum[threadIdx.x], in[ii]);        \
  }                                                             \
  __syncthreads();                                              \
  if (threadIdx.x == 0)                                         \
  {                                                             \
    float res = accum[0];                                       \
    int limit;                                                  \
    if (start+blockDim.x > N) limit=(N-start);  \
    else limit=blockDim.x;                                      \
    for (int ii = 1; ii < limit; ii++) {                  \
      res=OP(res,accum[ii]);                                    \
     }                                                          \
    out[blockIdx.x] = res;                                      \
  }                                                             \
}                                                               \
                                                                \
extern "C" const char * CUDA ## FktName(float * a, int N, float * resp) \
{                                                               \
  hipError_t myerr;                                            \
  const char * myerrStr;                                              \
  float res;                                                    \
  int CUIMAGE_REDUCE_BLOCKS=NBLOCKS(N,CUIMAGE_REDUCE_THREADS);  \
  dim3 threadBlock(CUIMAGE_REDUCE_THREADS);                     \
  dim3 blockGrid(CUIMAGE_REDUCE_BLOCKS);                        \
  myerrStr=CheckReduceAllocation(2*CUIMAGE_REDUCE_BLOCKS);      \
  if (myerrStr) return myerrStr;                                \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, TmpRedArray, N);       \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpy(accum, TmpRedArray, CUIMAGE_REDUCE_BLOCKS*sizeof(float), hipMemcpyDeviceToHost);\
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  res = accum[0];                                               \
  for (int ii=1; ii < CUIMAGE_REDUCE_BLOCKS; ii++)  {           \
    res=OP(res,accum[ii]);                                      \
   }                                                            \
  /* hipFree(TmpRedArray); */                                  \
  /* free(accum); */                                            \
                                                                \
  (* resp)=res;                                                 \
  return 0;                                                     \
}

// The version below is for complex valued arrays

#define CUDA_FullRedCpx(FktName, OP)               \
__global__ void FktName (float *in, float *out, int N){      \
  const int stride = blockDim.x * gridDim.x;                    \
  const int start  = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;\
  __shared__ float accum[CUIMAGE_REDUCE_THREADS];               \
  __shared__ float accumI[CUIMAGE_REDUCE_THREADS];              \
  if (start >= N) return;                                    \
                                                                \
  accum[threadIdx.x] = in[2*start];                             \
  accumI[threadIdx.x] = in[2*start+1];                          \
  for (int ii=start+stride; ii < N; ii += stride)  {         \
    accum[threadIdx.x] = OP(accum[threadIdx.x], in[2*ii]);      \
    accumI[threadIdx.x] = OP(accumI[threadIdx.x], in[2*ii +1]); \
  }                                                             \
  __syncthreads();                                              \
  if (!threadIdx.x)                                             \
  {                                                             \
    float res = accum[0];                                       \
    float resI = accumI[0];                                     \
    int limit;                                                  \
    if (start+blockDim.x > N) limit=(N-start);  \
    else limit=blockDim.x;                                      \
    for (int ii = 1; ii < limit; ii++) {                        \
      res=OP(res,accum[ii]);                                    \
      resI=OP(resI,accumI[ii]);                                 \
     }                                                          \
    out[2*blockIdx.x] = res;                                    \
    out[2*blockIdx.x + 1] = resI;                               \
  }                                                             \
}  \
\
extern "C" const char * CUDA ## FktName(float * a, int N, float * resp) \
{                                                               \
    hipError_t myerr;                                          \
  const char * myerrStr;                                              \
  float res, resI;                                              \
  int CUIMAGE_REDUCE_BLOCKS=NBLOCKS(N,CUIMAGE_REDUCE_THREADS);  \
  dim3 threadBlock(CUIMAGE_REDUCE_THREADS);                     \
  dim3 blockGrid(CUIMAGE_REDUCE_BLOCKS);                        \
  myerrStr=CheckReduceAllocation(2*CUIMAGE_REDUCE_BLOCKS);      \
  if (myerrStr) return myerrStr;                                \
                                                                \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, TmpRedArray, N);       \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpy(accum, TmpRedArray, 2*CUIMAGE_REDUCE_BLOCKS*sizeof(float), hipMemcpyDeviceToHost);\
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
                                                                \
  res = accum[0];                                               \
  resI = accum[1];                                              \
  for (int ii=1; ii < CUIMAGE_REDUCE_BLOCKS; ii++)  {           \
    res=OP(res,accum[2*ii]);                                    \
    resI=OP(resI,accum[2*ii + 1]);                              \
   }                                                            \
  /* hipFree(interm);  */                                      \
  /* free(accum); */                                            \
                                                                \
  (* resp)=res;                                                 \
  (* (resp+1))=resI;                                            \
  return 0;                                                     \
}

// The version below is for remembering the index (e.g. max and min)

#define CUDA_FullRedIdx(FktName, OP)               \
__global__ void FktName (float *in, float *out, int size){      \
  const int stride = blockDim.x * gridDim.x;                    \
  const int start  = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;\
  __shared__ float accum[CUIMAGE_REDUCE_THREADS];               \
  __shared__ float accumI[CUIMAGE_REDUCE_THREADS];              \
  if (start >= size) return;                                    \
                                                                \
  accum[threadIdx.x] = in[start];                               \
  accumI[threadIdx.x] = start;                                  \
  for (int ii=start+stride; ii < size; ii += stride)  {         \
    if OP(accum[threadIdx.x], in[ii]) { accum[threadIdx.x]= in[ii]; accumI[threadIdx.x]= ii; }      \
  }                                                             \
  __syncthreads();                                              \
  if (!threadIdx.x)                                             \
  {                                                             \
    float res = accum[0];                                       \
    float resI = accumI[0];                                     \
    int limit;                                                  \
    if (start+blockDim.x > size) limit=1+(size-start-1)/gridDim.x;  \
    else limit=blockDim.x;                                      \
    for (int ii = 1; ii < limit; ii++) {                        \
    if OP(res, accum[ii]){ res= accum[ii]; resI= accumI[ii]; }  \
     }                                                          \
    out[2*blockIdx.x] = res;                                    \
    out[2*blockIdx.x + 1] = resI;                               \
  }                                                             \
}  \
\
extern "C" const char * CUDA ## FktName(float * a, int N, float * resp) \
{                                                               \
  float res, resI;                                              \
  hipError_t myerr;                                            \
  const char * myerrStr;                                              \
  int CUIMAGE_REDUCE_BLOCKS=NBLOCKS(N,CUIMAGE_REDUCE_THREADS);  \
  dim3 threadBlock(CUIMAGE_REDUCE_THREADS);                     \
  dim3 blockGrid(CUIMAGE_REDUCE_BLOCKS);                        \
  myerrStr=CheckReduceAllocation(2*CUIMAGE_REDUCE_BLOCKS);      \
  if (myerrStr) return myerrStr;                                \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(a, TmpRedArray, N);       \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpy(accum, TmpRedArray, 2*CUIMAGE_REDUCE_BLOCKS*sizeof(float), hipMemcpyDeviceToHost);\
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  res = accum[0];                                               \
  resI = accum[1];                                              \
  for (int ii=1; ii < CUIMAGE_REDUCE_BLOCKS; ii++)  {           \
    if OP(res, accum[2*ii]) {res= accum[2*ii]; resI= accum[2*ii+1];  }  \
   }                                                            \
  /* hipFree(TmpRedArray); */                                  \
  /* free(accum); */                                            \
                                                                \
  (* resp)=res;                                                 \
  (* (resp+1))=resI;                                            \
  return 0;                                                     \
}


// Allows to work with the linear index image from a binary mask image.
// useful for: a(mask) = 2*a(mask)
// Algorithm: pass1 : count ones in your area
// pass 2: integrate accum over thread number to get block ones offset
// pass 3: Apply index

#define CUDA_MaskIdx(FktName, EXPRESSIONS)                      \
__global__ void FktName (float *a, float * mask,float *c, int N){ \
  int Blocksize = N/CUIMAGE_REDUCE_THREADS + 1;                 \
  int start = Blocksize * threadIdx.x;                          \
  __shared__ int accum[CUIMAGE_REDUCE_THREADS+1];               \
  if (start >= N) return;                                       \
                                                                \
  { int SumMask=0;                                              \
  for (int ii=start; ii < start+Blocksize; ii ++)  {            \
    if (ii < N)                                                 \
        SumMask += (mask[ii] != 0);                             \
  }                                                             \
  accum[threadIdx.x+1] = SumMask;                               \
  }                                                             \
  __syncthreads();                                              \
  if (threadIdx.x == 0)                                         \
  {                                                             \
    accum[0] = 0;                                               \
    int res = 0;                                                \
    for (int ii = 0; ii*Blocksize < N; ii++) {                  \
      res += accum[ii+1];                                       \
      accum[ii+1] = res;                                        \
     }                                                          \
    cuda_resultInt = res;                                       \
  }                                                             \
  __syncthreads();                                              \
  int mask_idx= accum[threadIdx.x];                             \
  for (int idx=start; idx < start+Blocksize; idx ++)  {         \
    if ((idx < N) && (mask[idx] != 0))                          \
      {                                                         \
        EXPRESSIONS                                             \
        mask_idx ++;                                            \
      }                                                         \
  }                                                             \
}                                                               \
                                                                \
extern "C" const char * CUDA ## FktName(float * in, float * mask, float *  out, int N, int * pM) \
{                                                               \
  int CUIMAGE_REDUCE_BLOCKS=1;                                  \
  hipError_t myerr;                                            \
  dim3 threadBlock(CUIMAGE_REDUCE_THREADS);                     \
  dim3 blockGrid(CUIMAGE_REDUCE_BLOCKS);                        \
                                                                \
  FktName<<<blockGrid, threadBlock>>>(in, mask, out, N);        \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
                                                                \
  hipMemcpyFromSymbol(pM, HIP_SYMBOL(cuda_resultInt), sizeof(* pM));\
                                                                \
  myerr=hipGetLastError();                                     \
  if (myerr != hipSuccess)                                     \
      return hipGetErrorString(myerr);                         \
  return 0;                                                     \
}



// In the expression one can use the variables idx (for real valued arrays) and idc (for complex valued arrays)
// -------------- caller function is also generated -------------
#define CUDA_BinaryFkt(FktName,expression)                          \
__global__ void                                                     \
FktName(float*a,float *b, float * c, int N)                         \
{                                                                   \
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float * b, float * c, int N)  \
{                                                                       \
    hipError_t myerr;                                          \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
	FktName<<<nBlocks,blockSize>>>(a,b,c,N);                            \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

// --------------Macro generating operation of array with real constant -------------

#define CUDA_UnaryFktConst(FktName,expression)                      \
__global__ void                                                     \
FktName(float*a,float b, float * c, int N)                          \
{                                                                   \
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float b, float * c, int N)  \
{                                                                       \
    hipError_t myerr;                                          \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
	FktName<<<nBlocks,blockSize>>>(a,b,c,N);                            \
    myerr=hipGetLastError();                                             \
    if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

// --------------Macro generating operation with complex array and constant -------------
#define CUDA_UnaryFktConstC(FktName,expression)                      \
__global__ void                                                     \
FktName(float*a,float br, float bi, float * c, int N)               \
{                                                                   \
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float br, float bi, float * c, int N)  \
{                                                                       \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
   hipError_t myerr;                                          \
	FktName<<<nBlocks,blockSize>>>(a,br,bi,c,N);                        \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                       

        

// ----------- Makro for function with an integer Vector ---- e.g.- for cyclic shifts etc. -----
#define CUDA_UnaryFktIntVec(FktName,expression)                      \
__global__ void                                                     \
FktName(float*a, SizeND b, float * c, SizeND sSize, int N)          \
{                                                                   \
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   \
    expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, int b[CUDA_MAXDIM], float * c, int mySize[CUDA_MAXDIM], int N)  \
{                                                                       \
  hipError_t myerr;                                          \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    SizeND sb,sSize;                                                    \
    for (int d=0;d<CUDA_MAXDIM;d++)                                     \
    { sb.s[d]=b[d];sSize.s[d]=mySize[d]; }                              \
	FktName<<<nBlocks,blockSize>>>(a,sb,c,sSize,N);                     \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

// ----------- Makro for function with an integer Vector ---- e.g.- for cyclic shifts etc. -----
#define CUDA_Fkt2Vec(FktName,expression)                            \
__global__ void                                                     \
FktName(float * c, VecND vec1, VecND vec2, SizeND sSize, int N)     \
{                                                                   \
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   \
    expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * c, VecND vec1, VecND vec2, SizeND sSize, int N)  \
{                                                                       \
    hipError_t myerr;                                          \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
	FktName<<<nBlocks,blockSize>>>(c,vec1,vec2,sSize,N);                \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}



// --------------Macro generating unary operation with complex array  -------------
#define CUDA_UnaryFkt(FktName,expression)                     \
__global__ void                                                     \
FktName(float*a, float * c, int N)                                  \
{                                                                   \
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   \
	expression                                                      \
}                                                                   \
extern "C" const char * CUDA ## FktName(float * a, float * c, int N)         \
{                                                                       \
    hipError_t myerr;                                          \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
	FktName<<<nBlocks,blockSize>>>(a,c,N);                              \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

// ---------------------- Some functions which know about x, and z position --------
// gets two sources and one destination, the two sources are assumed to have the same size
// sx,sy,sz : Source array sizes (total)
// sox,soy,soy : offsets
// ssx, ssy,ssz : source (or destination) subarray sizes
// dx,dy,dz: destination total array sizes
// dox,doy,doz : destination offsets

// THESE STRUCT DEVINITION ARE NEEDED, AS CUDA CANNOT DEAL CORRECTLY WITH FIXED LENGTH ARRAYS IN THE ARGUMENT
// ACCESING THEM WILL CAUSE A CRASH!
// HOWEVER, STRUCTS WITH THE ARRAY INSIDE ARE OK
typedef struct {
    int s[3];
} Size3D ;

// Line below is used as an add-on to the 3d function below in case 3d assignment is needed
#define GET3DIDD int idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]);

#define CUDA_3DFkt(FktName,expressions)                                  \
__global__ void                                                         \
FktName(float *a, float *c, Size3D sSize,Size3D dSize,Size3D sOffs, Size3D sROI, Size3D dOffs) \
{                                                                       \
  int idx=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  int N=sROI.s[0]*sROI.s[1]*sROI.s[2];                                        \
  if(idx>=N) return;                                                    \
  int x=(idx)%sROI.s[0];                                                  \
  int y=(idx/sROI.s[0])%sROI.s[1];                                          \
  int z=(idx/(sROI.s[0]*sROI.s[1]))%sROI.s[2];                                \
  int ids=x+sOffs.s[0]+sSize.s[0]*(y+sOffs.s[1])+sSize.s[0]*sSize.s[1]*(z+sOffs.s[2]);                               \
  expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * a, float *c, int sSize[3], int dSize[3], int sOffs[3], int sROI[3], int dOffs[3])  \
{                                                                       \
    hipError_t myerr;                                                \
    int N=sROI[0]*sROI[1]*sROI[2];                                      \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    Size3D sS,dS,sO,sR,dO;                                              \
    int d;                                                              \
    for (d=0;d<3;d++)                                                   \
        {sS.s[d]=sSize[d];dS.s[d]=dSize[d];sO.s[d]=sOffs[d];sR.s[d]=sROI[d];dO.s[d]=dOffs[d];} \
	FktName<<<nBlocks,blockSize>>>(a,c,sS,dS,sO,sR,dO); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                     

// --- macros for sub-assigning a block with vectors in each dimension -----
#define CUDA_3DAsgFkt(FktName,expressions)                                  \
__global__ void                                                         \
FktName(float *c, float br, float bi, Size3D dSize, Size3D dROI, Size3D dOffs) \
{                                                                       \
  int idx=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  int N=dROI.s[0]*dROI.s[1]*dROI.s[2];                                        \
  if(idx>=N) return;                                                    \
  int x=(idx)%dROI.s[0];                                               \
  int y=(idx/dROI.s[0])%dROI.s[1];                                    \
  int z=(idx/(dROI.s[0]*dROI.s[1]))%dROI.s[2];                       \
  int idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]);                               \
  expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * c, float br, float bi, int dSize[3], int dROI[3], int dOffs[3])  \
{                                                                       \
    hipError_t myerr;                                                \
    int N=dROI[0]*dROI[1]*dROI[2];                                      \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    Size3D dR,dS,dO;                                              \
    int d;                                                              \
    for (d=0;d<3;d++)                                                   \
        {dS.s[d]=dSize[d];dR.s[d]=dROI[d];dO.s[d]=dOffs[d];} \
	FktName<<<nBlocks,blockSize>>>(c,br,bi,dS,dR,dO); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                     

// --- macros for sub-assigning a block with vectors in each dimension - Extended version to be suitable for repmat
#define CUDA_3DWrapAsgFkt(FktName,expressions)                          \
__global__ void                                                         \
FktName(float *a, float *c, Size3D dSize, Size3D sSize)       \
{                                                                       \
  int idd=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  int N=dSize.s[0]*dSize.s[1]*dSize.s[2];                               \
  if(idd>=N) return;                                                    \
  int x=(idd)%dSize.s[0];                                                \
  int y=(idd/dSize.s[0])%dSize.s[1];                                    \
  int z=(idd/(dSize.s[0]*dSize.s[1]))%dSize.s[2];                       \
  int ids=x%sSize.s[0]+sSize.s[0]*(y%sSize.s[1])+sSize.s[0]*sSize.s[1]*(z%sSize.s[2]); \
  expressions                                                           \
}                                                                       \
extern "C" const char * CUDA ## FktName(float *a, float * c, int sSize[3], int dSize[3])  \
{                                                                       \
    hipError_t myerr;                                                \
    int N=dSize[0]*dSize[1]*dSize[2];                                      \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    Size3D sS,dS;                                              \
    int d;                                                              \
    for (d=0;d<3;d++)                                                   \
        {dS.s[d]=dSize[d];sS.s[d]=sSize[d];} \
	FktName<<<nBlocks,blockSize>>>(a,c,dS,sS); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

//  Now the 5D Versions of the same code

// THESE STRUCT DEVINITION ARE NEEDED, AS CUDA CANNOT DEAL CORRECTLY WITH FIXED LENGTH ARRAYS IN THE ARGUMENT
// ACCESING THEM WILL CAUSE A CRASH!
// HOWEVER, STRUCTS WITH THE ARRAY INSIDE ARE OK
typedef struct {
    int s[5];
} Size5D ;

// Line below is used as an add-on to the 3d function below in case 3d assignment is needed
#define GET5DIDD   int idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])+dSize.s[0]*dSize.s[1]*dSize.s[2]*(t+dOffs.s[3])+dSize.s[0]*dSize.s[1]*dSize.s[2]*dSize.s[3]*(e+dOffs.s[4]);

#define CUDA_5DFkt(FktName,expressions)                                 \
__global__ void                                                         \
FktName(float *a, float *c, Size5D sSize,Size5D dSize,Size5D sOffs, Size5D sROI, Size5D dOffs) \
{                                                                     \
  int idx=(blockIdx.x*blockDim.x+threadIdx.x);                        \
  int N=sROI.s[0]*sROI.s[1]*sROI.s[2]*sROI.s[3]*sROI.s[4];            \
  if(idx>=N) return;                                                  \
  int x=(idx)%sROI.s[0];                                              \
  int y=(idx/sROI.s[0])%sROI.s[1];                                    \
  int z=(idx/(sROI.s[0]*sROI.s[1]))%sROI.s[2];                        \
  int t=(idx/(sROI.s[0]*sROI.s[1]*sROI.s[2]))%sROI.s[3];              \
  int e=(idx/(sROI.s[0]*sROI.s[1]*sROI.s[2]*sROI.s[3]))%sROI.s[4];    \
  int ids=x+sOffs.s[0]+sSize.s[0]*(y+sOffs.s[1])+sSize.s[0]*sSize.s[1]*(z+sOffs.s[2])+sSize.s[0]*sSize.s[1]*sSize.s[2]*(t+sOffs.s[3])+sSize.s[0]*sSize.s[1]*sSize.s[2]*sSize.s[3]*(e+sOffs.s[4]);   \
  expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * a, float *c, int sSize[5], int dSize[5], int sOffs[5], int sROI[5], int dOffs[5])  \
{                                                                       \
    hipError_t myerr;                                                \
    int N=sROI[0]*sROI[1]*sROI[2]*sROI[3]*sROI[4];                      \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    Size5D sS,dS,sO,sR,dO;                                              \
    int d;                                                              \
    for (d=0;d<5;d++)                                                   \
        {sS.s[d]=sSize[d];dS.s[d]=dSize[d];sO.s[d]=sOffs[d];sR.s[d]=sROI[d];dO.s[d]=dOffs[d];} \
	FktName<<<nBlocks,blockSize>>>(a,c,sS,dS,sO,sR,dO); \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      
                
// --- macros for sub-assigning a block with vectors in each dimension -----

#define CUDA_5DAsgFkt(FktName,expressions)                                  \
__global__ void                                                         \
FktName(float *c, float br, float bi, Size5D dSize, Size5D dROI, Size5D dOffs) \
{                                                                       \
  int idx=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  int N=dROI.s[0]*dROI.s[1]*dROI.s[2]*dROI.s[3]*dROI.s[4];              \
  if(idx>=N) return;                                                    \
  int x=(idx)%dROI.s[0];                                               \
  int y=(idx/dROI.s[0])%dROI.s[1];                                    \
  int z=(idx/(dROI.s[0]*dROI.s[1]))%dROI.s[2];                       \
  int t=(idx/(dROI.s[0]*dROI.s[1]*dROI.s[2]))%dROI.s[3];              \
  int e=(idx/(dROI.s[0]*dROI.s[1]*dROI.s[2]*dROI.s[3]))%dROI.s[4];    \
  int idd=x+dOffs.s[0]+dSize.s[0]*(y+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])+dSize.s[0]*dSize.s[1]*dSize.s[2]*(t+dOffs.s[3])+dSize.s[0]*dSize.s[1]*dSize.s[2]*dSize.s[3]*(e+dOffs.s[4]);   \
  expressions                                                            \
}                                                                       \
extern "C" const char * CUDA ## FktName(float * c, float br, float bi, int dSize[5], int dROI[5], int dOffs[5])  \
{                                                                       \
    hipError_t myerr;                                                \
    int N=dROI[0]*dROI[1]*dROI[2]*dROI[3]*dROI[4];                      \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    Size5D dR,dS,dO;                                                    \
    int d;                                                              \
    for (d=0;d<5;d++)                                                   \
        {dS.s[d]=dSize[d];dR.s[d]=dROI[d];dO.s[d]=dOffs[d];}            \
	FktName<<<nBlocks,blockSize>>>(c,br,bi,dS,dR,dO);                   \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

// --- macros for sub-assigning a block with vectors in each dimension - Extended version to be suitable for repmat
#define CUDA_5DWrapAsgFkt(FktName,expressions)                          \
__global__ void                                                         \
FktName(float *a, float *c, Size5D dSize, Size5D sSize)       \
{                                                                       \
  int idd=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  int N=dSize.s[0]*dSize.s[1]*dSize.s[2]*dSize.s[3]*dSize.s[4];                               \
  if(idd>=N) return;                                                    \
  int x=(idd)%dSize.s[0];                                                \
  int y=(idd/dSize.s[0])%dSize.s[1];                                    \
  int z=(idd/(dSize.s[0]*dSize.s[1]))%dSize.s[2];                       \
  int t=(idd/(dSize.s[0]*dSize.s[1]*dSize.s[2]))%dSize.s[3];              \
  int e=(idd/(dSize.s[0]*dSize.s[1]*dSize.s[2]*dSize.s[3]))%dSize.s[4];    \
  int ids=x%sSize.s[0]+sSize.s[0]*(y%sSize.s[1])+sSize.s[0]*sSize.s[1]*(z%sSize.s[2]) + sSize.s[0]*sSize.s[1]*sSize.s[2]*(t%sSize.s[3])+sSize.s[0]*sSize.s[1]*sSize.s[2]*sSize.s[3]*(e%sSize.s[4]); \
  expressions                                                           \
}                                                                       \
extern "C" const char * CUDA ## FktName(float *a, float * c, int sSize[5], int dSize[5])  \
{                                                                       \
    hipError_t myerr;                                                  \
    int N=dSize[0]*dSize[1]*dSize[2]*dSize[3]*dSize[4];                 \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          \
    Size5D sS,dS;                                                       \
    int d;                                                              \
    for (d=0;d<5;d++)                                                   \
        {dS.s[d]=dSize[d];sS.s[d]=sSize[d];}                            \
	FktName<<<nBlocks,blockSize>>>(a,c,dS,sS);                          \
  myerr=hipGetLastError();                                             \
  if (myerr != hipSuccess)                                             \
      return hipGetErrorString(myerr);                                 \
  return 0;                                                             \
}                                                                      

// The funciton below checks whether the size of allocated reduce arrays is sufficient and reallocates if needed be
// The arrays are "accum" and "TmpRedArray"
const char * CheckReduceAllocation(int asize) {
    hipError_t myerr;
    asize=((asize/MinRedBlockSize) + 1)*MinRedBlockSize;  // round it up to the nearest multiple of MinRedSize
    if (! accum){
       accum = (float *) malloc(asize*sizeof(float));
       if (! accum)
       return "CheckReduceAllocation: Malloc failed";
    }    
    if (! TmpRedArray) {
        hipMalloc((void **) &TmpRedArray, asize*sizeof(float));
        CurrentRedSize=asize;
        myerr=hipGetLastError();
        if (myerr != hipSuccess)
          return hipGetErrorString(myerr);
    }
    
    if (asize > CurrentRedSize)
    {
        free(accum);
        accum = (float *) malloc(asize*sizeof(float));
        if (! accum)
            return "CheckReduceAllocation: ReMalloc failed";
        hipFree(TmpRedArray);
        myerr=hipGetLastError();
        if (myerr != hipSuccess)
            return hipGetErrorString(myerr);

        hipMalloc((void **) &TmpRedArray, asize*sizeof(float));
        myerr=hipGetLastError();
        if (myerr != hipSuccess)
            return hipGetErrorString(myerr);
        CurrentRedSize=asize;
    }
    return 0;
}

extern "C" int GetCurrentRedSize(void) {
    return CurrentRedSize;
}

/*__global__ void                                                         \
bla_ ## FktName(float*a, float * c, int N,  Size3D sSize,Size3D dSize,Size3D sOffs, Size3D sROI, Size3D dOffs) {                                    \
  int idx=(blockIdx.x*blockDim.x+threadIdx.x);                          \
  int idcd=0,idcs=0,ids=0;                                                    \
  if(idx>=N) return;                                                    \
    expression                                                          \
}   \  */

//	FktName<<<nBlocks,blockSize>>>(a,c,sSize,dSize,sOffs, sROI, dOffs); \


CUDA_FullRed(sum_arr,mysum)
//CUDA_FullRedBin(sum_arr,mysum)
CUDA_FullRedCpx(sum_carr,mysum)
// CUDA_FullRed(sum_carr,res+=accum[ii];)
CUDA_FullRedIdx(max_arr,maxCond)
CUDA_FullRedIdx(min_arr,minCond)

CUDA_PartRedMask(psum_arr,mysum)
CUDA_PartRedMaskCpx(psum_carr,mysum)
CUDA_PartRedMaskIdx(pmax_arr,maxCond)
CUDA_PartRedMaskIdx(pmin_arr,minCond)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// CUDA_3DFkt(arr_subcpy_arr,c[idd]=a[ids];)
CUDA_3DAsgFkt(const_3dsubcpy_arr,c[idd]=br;)
CUDA_3DAsgFkt(cconst_3dsubcpy_carr,c[2*idd]=br;c[2*idd+1]=bi;)

// repcopy for repmat command
CUDA_3DWrapAsgFkt(arr_3drepcpy_arr,c[idd]=a[ids];)
CUDA_3DWrapAsgFkt(carr_3drepcpy_carr,c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign

CUDA_5DAsgFkt(const_5dsubcpy_arr,c[idd]=br;)
CUDA_5DAsgFkt(cconst_5dsubcpy_carr,c[2*idd]=br;c[2*idd+1]=bi;)

// repcopy for repmat command
CUDA_5DWrapAsgFkt(arr_5drepcpy_arr,c[idd]=a[ids];)
CUDA_5DWrapAsgFkt(carr_5drepcpy_carr,c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)

// Assigning constant values to arrays accessed with a boolean array
CUDA_UnaryFktConst(arr_boolassign_const,if (a[idx]!=0) c[idx]=b;)

CUDA_UnaryFktConstC(carr_boolassign_const,if (a[idx]!=0) {c[2*idx]=br;c[2*idx+1]=bi;})

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// CUDA_3DFkt(arr_subcpy_arr,c[idd]=a[ids];)
CUDA_3DFkt(arr_3dsubcpy_arr, GET3DIDD; c[idd]=a[ids];)
CUDA_3DFkt(carr_3dsubcpy_carr, GET3DIDD; c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)
CUDA_3DFkt(arr_3dsubcpy_carr, GET3DIDD; c[2*idd]=a[ids];c[2*idd+1]=0;)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// These versions intoduce a transpose operation
CUDA_3DFkt(arr_3dsubcpyT_arr,  int iddt=y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]); c[iddt]=a[ids];)
CUDA_3DFkt(carr_3dsubcpyT_carr,int idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=a[2*ids+1];)
// with conjugation
CUDA_3DFkt(carr_3dsubcpyCT_carr,int idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=-a[2*ids+1];)

//CUDA_3DFkt(arr_subref_arr3d,c[idd]=)
//getCudaRef(prhs[1]),newarr,sSize,dSize,cuda_array[newref[0]],cuda_array[newref[1]],cuda_array[newref[2]]);

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
CUDA_5DFkt(arr_5dsubcpy_arr, GET5DIDD; c[idd]=a[ids];)
CUDA_5DFkt(carr_5dsubcpy_carr, GET5DIDD; c[2*idd]=a[2*ids];c[2*idd+1]=a[2*ids+1];)
CUDA_5DFkt(arr_5dsubcpy_carr, GET5DIDD; c[2*idd]=a[ids];c[2*idd+1]=0;)

// Sub copying, copies a source area into a destination area. Can be used for cat and subassign
// These versions intoduce a transpose operation
CUDA_5DFkt(arr_5dsubcpyT_arr,  int iddt=y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2]); c[iddt]=a[ids];)
CUDA_5DFkt(carr_5dsubcpyT_carr,int idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=a[2*ids+1];)
 // with conjugation
CUDA_5DFkt(carr_5dsubcpyCT_carr,int idcdt=2*(y+dOffs.s[0]+dSize.s[0]*(x+dOffs.s[1])+dSize.s[0]*dSize.s[1]*(z+dOffs.s[2])); c[idcdt]=a[2*ids];c[idcdt+1]=-a[2*ids+1];)  


// Power
CUDA_BinaryFkt(arr_power_arr,c[idx]=pow(a[idx],b[idx]);)
CUDA_UnaryFktConst(arr_power_const,c[idx]=pow(a[idx],b);)
CUDA_UnaryFktConst(const_power_arr,c[idx]=pow(b,a[idx]);)

// Multiplications
CUDA_BinaryFkt(arr_times_arr,c[idx]=a[idx]*b[idx];)
CUDA_BinaryFkt(carr_times_carr,
    int idc=2*idx;
    float myr=a[idc]*b[idc]-a[idc+1]*b[idc+1];float myi=a[idc]*b[idc+1]+a[idc+1]*b[idc];
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_BinaryFkt(arr_times_carr,c[2*idx]=a[idx]*b[2*idx];c[2*idx+1]=a[idx]*b[2*idx+1];)
CUDA_BinaryFkt(carr_times_arr,c[2*idx]=a[2*idx]*b[idx];c[2*idx+1]=a[2*idx+1]*b[idx];)
//CUDA_BinaryFkt(arr_times_carr,c[2*idx]=a[idx]*b[2*idx];c[2*idx+1]=a[idx+1]*b[2*idx];)
CUDA_UnaryFktConst(arr_times_const,c[idx]=a[idx]*b;)
CUDA_UnaryFktConst(const_times_arr,c[idx]=a[idx]*b;)
CUDA_UnaryFktConstC(carr_times_const,
    int idc=2*idx;
    float myr=a[idc]*br-a[idc+1]*bi;float myi=a[idc]*bi+a[idc+1]*br;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(const_times_carr,
    int idc=2*idx;
    float myr=a[idc]*br-a[idc+1]*bi;float myi=a[idc]*bi+a[idc+1]*br;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(arr_times_Cconst,c[2*idx]=a[idx]*br;c[2*idx+1]=a[idx]*bi;)
CUDA_UnaryFktConstC(Cconst_times_arr,c[2*idx]=br*a[idx];c[2*idx+1]=bi*a[idx];)

// Divisions
CUDA_BinaryFkt(arr_divide_arr,c[idx]=a[idx]/b[idx];)
CUDA_BinaryFkt(carr_divide_carr,
    int idc=2*idx;
    float tmp=b[idc]*b[idc]+b[idc+1]*b[idc+1];
    float myr=(a[idc]*b[idc]+a[idc+1]*b[idc+1])/tmp;float myi=(a[idc+1]*b[idc]-a[idc]*b[idc+1])/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_BinaryFkt(carr_divide_arr,c[2*idx]=a[2*idx]/b[idx];c[2*idx+1]=a[2*idx+1]/b[idx];)
CUDA_BinaryFkt(arr_divide_carr,
    int idc=2*idx;
    float tmp=b[idc]*b[idc]+b[idc+1]*b[idc+1];
    float myr=(a[idx]*b[idc]+a[idx+1]*b[idc+1])/tmp;float myi=(a[idx+1]*b[idc]-a[idx]*b[idc+1])/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConst(arr_divide_const,c[idx]=a[idx]/b;)
CUDA_UnaryFktConst(const_divide_arr,c[idx]=b/a[idx];)
CUDA_UnaryFktConstC(carr_divide_const,
    int idc=2*idx;
    float tmp=br*br+bi*bi;
    float myr=(a[idc]*br+a[idc+1]*bi)/tmp;float myi=(a[idc+1]*br-a[idc]*bi)/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(const_divide_carr,
    int idc=2*idx;
    float tmp=a[idc]*a[idc]+a[idc+1]*a[idc+1];
    float myr=(br*a[idc]+bi*a[idc+1])/tmp;float myi=(bi*a[idc]-br*a[idc+1])/tmp;
    c[idc]=myr;c[idc+1]=myi;
)
CUDA_UnaryFktConstC(arr_divide_Cconst,
    float tmp=br*br+bi*bi;
    float myr=a[idx]*br/tmp;float myi= -a[idx]*bi/tmp;
    c[2*idx]=myr;c[2*idx+1]=myi;
)
CUDA_UnaryFktConstC(Cconst_divide_arr,c[2*idx]=br/a[idx];c[2*idx+1]=bi/a[idx];)

// Element-wise maximum operations
CUDA_BinaryFkt(arr_max_arr,c[idx]=a[idx]>b[idx]?a[idx]:b[idx];)
CUDA_BinaryFkt(carr_max_carr, int idc=2*idx; if (a[idc]*a[idc]+a[idc+1]*a[idc+1] > b[idc]*b[idc]+b[idc+1]*b[idc+1]) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=b[idc];c[idc+1]=b[idc+1];})
CUDA_BinaryFkt(carr_max_arr,int idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] > b[idx]*b[idx]) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=b[idx];c[idc+1]=0;})
CUDA_BinaryFkt(arr_max_carr,int idc=2*idx;if (a[idx]*a[idx] > b[idc]*b[idc]+b[idc+1]*b[idc+1]) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=b[idc];c[idc+1]=b[idc+1];})
CUDA_UnaryFktConst(arr_max_const,c[idx]=a[idx]>b?a[idx]:b;)
CUDA_UnaryFktConst(const_max_arr,c[idx]=a[idx]>b?a[idx]:b;)
CUDA_UnaryFktConstC(carr_max_const,int idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] > br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(const_max_carr,int idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] > br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(arr_max_Cconst,int idc=2*idx;if (a[idx]*a[idx] > br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(Cconst_max_arr,int idc=2*idx;if (a[idx]*a[idx] > br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})

// Element-wise minimum operations
CUDA_BinaryFkt(arr_min_arr,c[idx]=a[idx]<b[idx]?a[idx]:b[idx];)
CUDA_BinaryFkt(carr_min_carr, int idc=2*idx; if (a[idc]*a[idc]+a[idc+1]*a[idc+1] < b[idc]*b[idc]+b[idc+1]*b[idc+1]) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=b[idc];c[idc+1]=b[idc+1];})
CUDA_BinaryFkt(carr_min_arr,int idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] < b[idx]*b[idx]) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=b[idx];c[idc+1]=0;})
CUDA_BinaryFkt(arr_min_carr,int idc=2*idx;if (a[idx]*a[idx] < b[idc]*b[idc]+b[idc+1]*b[idc+1]) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=b[idc];c[idc+1]=b[idc+1];})
CUDA_UnaryFktConst(arr_min_const,c[idx]=a[idx]<b?a[idx]:b;)
CUDA_UnaryFktConst(const_min_arr,c[idx]=a[idx]<b?a[idx]:b;)
CUDA_UnaryFktConstC(carr_min_const,int idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] < br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(const_min_carr,int idc=2*idx;if (a[idc]*a[idc]+a[idc+1]*a[idc+1] < br*br+bi*bi) {c[idc]=a[idc];c[idc+1]=a[idc+1];}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(arr_min_Cconst,int idc=2*idx;if (a[idx]*a[idx] < br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})
CUDA_UnaryFktConstC(Cconst_min_arr,int idc=2*idx;if (a[idx]*a[idx] < br*br+bi*bi) {c[idc]=a[idx];c[idc+1]=0;}else{ c[idc]=br;c[idc+1]=bi;})

// Additions
CUDA_BinaryFkt(arr_plus_arr,c[idx]=a[idx]+b[idx];)
CUDA_BinaryFkt(carr_plus_carr, int idc=2*idx; c[idc]=a[idc]+b[idc];c[idc+1]=a[idc+1]+b[idc+1];)
CUDA_BinaryFkt(carr_plus_arr,int idc=2*idx;c[idc]=a[idc]+b[idx];c[idc+1]=a[idc+1];)
CUDA_BinaryFkt(arr_plus_carr,int idc=2*idx;c[idc]=a[idx]+b[idc];c[idc+1]=b[idc+1];)
CUDA_UnaryFktConst(arr_plus_const,c[idx]=a[idx]+b;)
CUDA_UnaryFktConst(const_plus_arr,c[idx]=a[idx]+b;)
CUDA_UnaryFktConstC(carr_plus_const,int idc=2*idx;c[idc]=a[idc]+br;c[idc+1]=a[idc+1]+bi;)
CUDA_UnaryFktConstC(const_plus_carr,int idc=2*idx;c[idc]=a[idc]+br;c[idc+1]=a[idc+1]+bi;)
CUDA_UnaryFktConstC(arr_plus_Cconst,int idc=2*idx;c[idc]=a[idx]+br;c[idc+1]=bi;)
CUDA_UnaryFktConstC(Cconst_plus_arr,int idc=2*idx;c[idc]=br+a[idx];c[idc+1]=bi;)

// Subtractions
CUDA_BinaryFkt(arr_minus_arr,c[idx]=a[idx]-b[idx];)
CUDA_BinaryFkt(carr_minus_carr,int idc=2*idx; c[idc]=a[idc]-b[idc];c[idc+1]=a[idc+1]-b[idc+1];)
CUDA_BinaryFkt(carr_minus_arr,int idc=2*idx;c[idc]=a[idc]-b[idx];c[idc+1]=a[idc+1];)
CUDA_BinaryFkt(arr_minus_carr,int idc=2*idx;c[idc]=a[idx]-b[idc];c[idc+1]=-b[idc+1];)
CUDA_UnaryFktConst(arr_minus_const,c[idx]=a[idx]-b;)
CUDA_UnaryFktConst(const_minus_arr,c[idx]=b-a[idx];)
CUDA_UnaryFktConstC(carr_minus_const,int idc=2*idx;c[idc]=a[idc]-br;c[idc+1]=a[idc+1]-bi;)
CUDA_UnaryFktConstC(const_minus_carr,int idc=2*idx;c[idc]=br-a[idc];c[idc+1]=bi-a[idc+1];)
CUDA_UnaryFktConstC(arr_minus_Cconst,int idc=2*idx;c[idc]=a[idx]-br;c[idc+1]=a[idx]-bi;)
CUDA_UnaryFktConstC(Cconst_minus_arr,int idc=2*idx;c[idc]=br-a[idx];c[idc+1]=bi;)

// Referencing and assignment  // STILL NEEDS SOME WORK
// CUDA_BinaryFkt(arr_subsref_arr,c[idx]=(b[idx] == 0) ? 0 : a[idx];)
// CUDA_BinaryFkt(carr_subsref_arr,c[idc]=(b[idx] == 0) ? 0 : a[idc]; c[idc+1]=(b[idx] == 0) ? 0 : a[idc+1];)
// CUDA_BinaryFkt(arr_subsasgn_arr,if (b[idx] == 0) c[idx] = a[idx];)
// CUDA_BinaryFkt(carr_subsasgn_arr,if (b[idx] == 0) {c[idc] = a[idc];c[idc+1] = a[idc+1];})
CUDA_MaskIdx(arr_subsref_arr,c[mask_idx]=a[idx];)
CUDA_MaskIdx(carr_subsref_arr,c[2*mask_idx]=a[2*idx]; c[2*mask_idx+1]=a[2*idx+1];)
CUDA_MaskIdx(arr_subsasgn_arr,a[idx]=c[mask_idx];)
CUDA_MaskIdx(carr_subsasgn_arr,a[2*idx]=c[2*mask_idx]; a[2*idx+1]=c[2*mask_idx+1];)

// diagonal matrix generation
CUDA_3DFkt(arr_diag_set,  int iddt=ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1]); c[iddt]=a[ids];)
CUDA_3DFkt(carr_diag_set,  int idcdt=2*(ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1])); c[idcdt]=a[2*ids];c[idcdt+1]=a[2*ids*1];)
CUDA_3DFkt(arr_diag_get,  int iddt=ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1]); a[ids]=c[iddt];)
CUDA_3DFkt(carr_diag_get,  int idcdt=2*(ids+dOffs.s[0]+dSize.s[0]*(ids+dOffs.s[1])); a[2*ids]=c[idcdt];a[2*ids*1]=c[idcdt+1];)

// referencing and assignment with index vectors.No Index checking performed
CUDA_BinaryFkt(arr_subsref_vec,{c[idx]=a[(int) b[idx]];})
CUDA_BinaryFkt(carr_subsref_vec,{c[2*idx]=a[2*((int) b[idx])];c[2*idx+1]=a[2*((int) b[idx])+1];})

CUDA_BinaryFkt(arr_subsasg_vec,{c[(int) b[idx]]=a[idx];})
CUDA_BinaryFkt(carr_subsasg_vec,{c[2*((int) b[idx])]=a[2*idx];c[2*((int) b[idx])+1]=a[2*idx+1];})

// binary logical operations

CUDA_BinaryFkt(arr_or_arr,{c[idx]=(float) (a[idx]!=0) || (b[idx]!=0);})
CUDA_UnaryFktConst(arr_or_const,{c[idx]=(float) (a[idx]!=0) || (b!=0);})
CUDA_UnaryFktConst(const_or_arr,{c[idx]=(float) (b!=0) || (a[idx]!=0);})

CUDA_BinaryFkt(arr_and_arr,{c[idx]=(float) (a[idx]!=0) && (b[idx]!=0);})
CUDA_UnaryFktConst(arr_and_const,{c[idx]=(float) (a[idx]!=0) && (b!=0);})
CUDA_UnaryFktConst(const_and_arr,{c[idx]=(float) (b!=0) && (a[idx]!=0);})

// Unary logical operations
CUDA_UnaryFkt(not_arr,c[idx]=(a[idx] == 0);)

// Comparison
CUDA_BinaryFkt(arr_smaller_arr,c[idx]=a[idx]<b[idx];)
CUDA_UnaryFktConst(arr_smaller_const,c[idx]=a[idx]<b;)
CUDA_UnaryFktConst(const_smaller_arr,c[idx]=b<a[idx];)

CUDA_BinaryFkt(arr_larger_arr,c[idx]=a[idx]>b[idx];)
CUDA_UnaryFktConst(arr_larger_const,c[idx]=a[idx]>b;)
CUDA_UnaryFktConst(const_larger_arr,c[idx]=b>a[idx];)

CUDA_BinaryFkt(arr_smallerequal_arr,c[idx]=a[idx]<=b[idx];)
CUDA_UnaryFktConst(arr_smallerequal_const,c[idx]=a[idx]<=b;)
CUDA_UnaryFktConst(const_smallerequal_arr,c[idx]=b<=a[idx];)

CUDA_BinaryFkt(arr_largerequal_arr,c[idx]=a[idx]>=b[idx];)
CUDA_UnaryFktConst(arr_largerequal_const,c[idx]=a[idx]>=b;)
CUDA_UnaryFktConst(const_largerequal_arr,c[idx]=b>=a[idx];)

// equals will always output a real valued array
CUDA_BinaryFkt(arr_equals_arr,c[idx]=(a[idx]==b[idx]);)
CUDA_BinaryFkt(carr_equals_carr, int idc=2*idx; c[idx]=(a[idc]==b[idc]) && (a[idc+1]==b[idc+1]);)
CUDA_BinaryFkt(carr_equals_arr,int idc=2*idx; c[idx]=(a[idc]==b[idx]) && (a[idc+1] == 0);)
CUDA_BinaryFkt(arr_equals_carr,int idc=2*idx; c[idx]=(a[idx]==b[idc]) && (b[idc+1] == 0);)
CUDA_UnaryFktConst(arr_equals_const,c[idx]=(a[idx]==b);)
CUDA_UnaryFktConst(const_equals_arr,c[idx]=(b==a[idx]);)
CUDA_UnaryFktConstC(carr_equals_const,int idc=2*idx; c[idx]=(a[idc]==br) && (a[idc+1]==bi);)
CUDA_UnaryFktConstC(const_equals_carr,int idc=2*idx; c[idx]=(br==a[idc]) && (bi==a[idc+1]);)
CUDA_UnaryFktConstC(arr_equals_Cconst,c[idx]=(a[idx]==br) && (bi==0);)
CUDA_UnaryFktConstC(Cconst_equals_arr,c[idx]=(br==a[idx]) && (bi==0);)

// not equals will always output a real valued array
CUDA_BinaryFkt(arr_unequals_arr,c[idx]=(a[idx]!=b[idx]);)
CUDA_BinaryFkt(carr_unequals_carr, int idc=2*idx; c[idx]=(a[idc]!=b[idc]) || (a[idc+1]!=b[idc+1]);)
CUDA_BinaryFkt(carr_unequals_arr,c[idx]=(a[2*idx]!=b[idx]) || (a[2*idx+1] != 0);)
CUDA_BinaryFkt(arr_unequals_carr,c[idx]=(a[idx]!=b[2*idx]) || (b[2*idx+1] != 0);)
CUDA_UnaryFktConst(arr_unequals_const,c[idx]=(a[idx]!=b);)
CUDA_UnaryFktConst(const_unequals_arr,c[idx]=(b!=a[idx]);)
CUDA_UnaryFktConstC(carr_unequals_const,c[idx]=(a[2*idx]!=br) || (a[2*idx+1]!=bi);)
CUDA_UnaryFktConstC(const_unequals_carr,c[idx]=(br!=a[2*idx]) || (bi!=a[2*idx+1]);)
CUDA_UnaryFktConstC(arr_unequals_Cconst,c[idx]=(a[idx]!=br) || (bi!=0);)
CUDA_UnaryFktConstC(Cconst_unequals_arr,c[idx]=(br!=a[idx]) || (bi!=0);)

// other Unary oparations
CUDA_UnaryFkt(uminus_arr,c[idx]=-a[idx];)
CUDA_UnaryFkt(uminus_carr,int idc=2*idx; c[idc]=-a[idc];c[idc+1]=-a[idc+1];)   // negates real and imaginary part

CUDA_UnaryFkt(exp_arr,c[idx]= exp(a[idx]);)
CUDA_UnaryFkt(exp_carr,int idc=2*idx; float len=exp(a[idc]);c[idc]=len*cos(a[idc+1]);c[idc+1]=len*sin(a[idc+1]);)

CUDA_UnaryFkt(sin_arr,c[idx]= sin(a[idx]);)
CUDA_UnaryFkt(sin_carr,int idc=2*idx; c[idc]=sin(a[idc])*cosh(a[idc+1]);c[idc+1]=cos(a[idc])*sinh(a[idc+1]);)

CUDA_UnaryFkt(cos_arr,c[idx]= cos(a[idx]);)
CUDA_UnaryFkt(cos_carr,int idc=2*idx; c[idc]=cos(a[idc])*cosh(a[idc+1]);c[idc+1]=sin(a[idc])*sinh(a[idc+1]);)

CUDA_UnaryFkt(sinh_arr,c[idx]= sinh(a[idx]);)
CUDA_UnaryFkt(sinh_carr,int idc=2*idx; c[idc]=sinh(a[idc])*cos(a[idc+1]);c[idc+1]=cosh(a[idc])*sin(a[idc+1]);)

CUDA_UnaryFkt(cosh_arr,c[idx]= cosh(a[idx]);)
CUDA_UnaryFkt(cosh_carr,int idc=2*idx; c[idc]=cosh(a[idc])*cos(a[idc+1]);c[idc+1]=sinh(a[idc])*sin(a[idc+1]);)

CUDA_UnaryFkt(sinc_arr, c[idx]= (a[idx] != 0) ? sin(a[idx])/a[idx] : 1.0;)
CUDA_UnaryFkt(sinc_carr,int idc=2*idx; c[idc]=0;c[idc+1]=0;) 
// c[idc]= (a[idc] == 0) ? sin(a[idc])*cosh(a[idc+1])/a[idc] : cosh(a[idc+1]);c[idc+1]= (a[idc] == 0) ? cos(a[idc])*sinh(a[idc+1])/a[idc] : sinh(a[idc+1]);)

CUDA_UnaryFkt(log_arr,c[idx]=log(a[idx]);)
CUDA_UnaryFkt(log_carr,c[2*idx]=log(a[2*idx]);c[2*idx+1]=0;)   //  not implemented

CUDA_UnaryFkt(abs_arr,c[idx]= (a[idx] > 0) ? a[idx] : -a[idx];)
CUDA_UnaryFkt(abs_carr,int idc=2*idx; c[idx]=sqrt(a[idc]*a[idc]+a[idc+1]*a[idc+1]);)

CUDA_UnaryFkt(conj_arr,c[idx]=a[idx];)
CUDA_UnaryFkt(conj_carr,int idc=2*idx; c[idc]=a[idc];c[idc+1]=-a[idc+1];)  // only affects the imaginary part

CUDA_UnaryFkt(sqrt_arr,c[idx]= sqrt(a[idx]);)
// funny expression below is the sign function ((x>0)-(x<0))
CUDA_UnaryFkt(sqrt_carr,int idc=2*idx; float L=sqrt(a[idc]*a[idc]+a[idc+1]*a[idc+1]); c[idc]=sqrt((L+a[idc])/2);c[idc+1]=((a[idc+1]>0)-(a[idc+1])<0)*sqrt((L-a[idc])/2);)

// Unary functions resulting in just a single value
CUDA_UnaryFkt(isIllegal_arr,if (isnan(a[idx]) || isinf(a[idx]) ) c[0]=1;)
CUDA_UnaryFkt(isIllegal_carr,if (a[2*idx+1]!=0 || isnan(a[2*idx]) || isnan(a[2*idx+1]) || isinf(a[2*idx]) || isinf(a[2*idx+1]) ) c[0]=1;)

CUDA_UnaryFkt(any_arr,if (a[idx]!=0) c[0]=1;)
CUDA_UnaryFkt(any_carr,if (a[2*idx]!=0 || a[2*idx+1]!=0) c[0]=1;)

// Binary functions with real valued input returning always complex arrays
CUDA_BinaryFkt(arr_complex_arr,c[2*idx]=a[idx];c[2*idx+1]=b[idx];)
CUDA_UnaryFktConst(arr_complex_const,c[2*idx]=a[idx];c[2*idx+1]=b;)
CUDA_UnaryFktConst(const_complex_arr,c[2*idx]=b;c[2*idx+1]=a[idx];)

// unary functions returning always real valued arrays

CUDA_UnaryFkt(real_arr,c[idx]=a[idx];)
CUDA_UnaryFkt(real_carr,c[idx]=a[2*idx];)

CUDA_UnaryFkt(imag_arr,c[idx]=0;)
CUDA_UnaryFkt(imag_carr,c[idx]=a[2*idx+1];)

CUDA_UnaryFkt(phase_arr,c[idx]=0;)
CUDA_UnaryFkt(phase_carr,c[idx]=atan2(a[2*idx+1],a[2*idx]);)

CUDA_UnaryFkt(isnan_arr,c[idx]=(float) isnan(a[idx]);)
CUDA_UnaryFkt(isnan_carr,c[idx]=(float) (isnan(a[2*idx])||isnan(a[2*idx+1]));)   // is not a number

CUDA_UnaryFkt(isinf_arr,c[idx]=(float) isinf(a[idx]);)
CUDA_UnaryFkt(isinf_carr,c[idx]=(float) (isinf(a[2*idx])||isinf(a[2*idx+1]));)   // is infinite


CUDA_UnaryFktIntVec(arr_circshift_vec,CoordsNDFromIdx(idx,sSize,pos);for(int _d=0;_d<CUDA_MAXDIM;_d++){pos.s[_d]-=b.s[_d];}int ids=0;IdxNDFromCoords(pos,sSize,ids);c[idx]=a[ids];)  // a[idx]
CUDA_UnaryFktIntVec(carr_circshift_vec,CoordsNDFromIdx(idx,sSize,pos);for(int _d=0;_d<CUDA_MAXDIM;_d++){pos.s[_d]-=b.s[_d];}int ids=0;IdxNDFromCoords(pos,sSize,ids);c[2*idx]=a[2*ids];c[2*idx+1]=a[2*ids+1];)

// In code below, the loop runs over the source dimensions. The array sizes are still set to the source sizes and will be (again) adjusted later
CUDA_UnaryFktIntVec(arr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos);
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=0;}
        for(_d=0;_d<CUDA_MAXDIM;_d++){
                if ((b.s[_d]<CUDA_MAXDIM) && (b.s[_d]>=0)) {
                        dSize.s[_d]=sSize.s[b.s[_d]]; posnew.s[_d] = pos.s[b.s[_d]];}
                } 
        int idd=0;IdxNDFromCoords(posnew,dSize,idd);c[idd]=a[idx];}) // a[idx]

CUDA_UnaryFktIntVec(carr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos);
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=0;}
        for(_d=0;_d<CUDA_MAXDIM;_d++){
                if ((b.s[_d]<CUDA_MAXDIM) && (b.s[_d]>=0)) {
                        dSize.s[_d]=sSize.s[b.s[_d]]; posnew.s[_d] = pos.s[b.s[_d]];}
                }
        int idd=0;IdxNDFromCoords(posnew,dSize,idd);c[2*idd]=a[2*idx];c[2*idd+1]=a[2*idx+1];}) 
/*
CUDA_UnaryFktIntVec(arr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos); \
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=pos.s[_d];} \
        for(_d=0;_d<CUDA_MAXDIM;_d++){ \
                if (b.s[_d]<CUDA_MAXDIM && b.s[_d]>=0) { \
                        dSize.s[b.s[_d]]=sSize.s[_d]; posnew.s[b.s[_d]] = pos.s[_d];} \
                } \
        int idd=0;IdxNDFromCoords(posnew,dSize,idd);c[idd]=a[idx];}) 

CUDA_UnaryFktIntVec(carr_permute_vec,{int _d;SizeND posnew; SizeND dSize; CoordsNDFromIdx(idx,sSize,pos); \
        for(_d=0;_d<CUDA_MAXDIM;_d++) {dSize.s[_d]=1;posnew.s[_d]=pos.s[_d];} \
        for(_d=0;_d<CUDA_MAXDIM;_d++){ \
                if (b.s[_d]<CUDA_MAXDIM && b.s[_d]>=0) { \
                        dSize.s[b.s[_d]]=sSize.s[_d]; posnew.s[b.s[_d]] = pos.s[_d];} \
                } \
        int idd=0;IdxNDFromCoords(posnew,dSize,idd);c[2*idd]=a[2*idx];c[2*idd+1]=a[2*idx+1];}) 
*/
        
CUDA_Fkt2Vec(arr_xyz_2vec,CoordsNDFromIdx(idx,sSize,pos);float val=0;for(int _d=0;_d<CUDA_MAXDIM;_d++){val += vec1.s[_d]+pos.s[_d]*(vec2.s[_d]-vec1.s[_d])/sSize.s[_d];} c[idx]=val;)  // a[idx]
CUDA_Fkt2Vec(arr_rr_2vec,CoordsNDFromIdx(idx,sSize,pos);float val=0;for(int _d=0;_d<CUDA_MAXDIM;_d++){val += Sqr(vec1.s[_d]+pos.s[_d]*(vec2.s[_d]-vec1.s[_d])/sSize.s[_d]);} c[idx]=sqrt(val);)  // a[idx]
CUDA_Fkt2Vec(arr_phiphi_2vec,CoordsNDFromIdx(idx,sSize,pos); c[idx]=atan2(vec1.s[0]+pos.s[0]*(vec2.s[0]-vec1.s[0])/sSize.s[0],vec1.s[1]+pos.s[1]*(vec2.s[1]-vec1.s[1])/sSize.s[1]);)  // phiphi

// Now include all the user-defined functions
// #include "user/user_cu_code.inc"
#include "user_cu_code.inc"


__global__ void set_arr(float b, float * c, int N)                          
{                                                                   
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   
	c[idx]=b;
}                                                                   
extern "C" const char * CUDAset_arr(float b, float * c, int N)  
{                                                                       
    hipError_t myerr;                                                \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          
	set_arr<<<nBlocks,blockSize>>>(b,c,N);                            
  myerr=hipGetLastError();                                             
  if (myerr != hipSuccess)                                             
      return hipGetErrorString(myerr);                                 
  return 0;                                                                   
}                                                                       

__global__ void set_carr(float br, float bi, float * c, int N)               
{                                                                   
	int idx=blockIdx.x*blockDim.x+threadIdx.x; if(idx>=N) return;   
    int idc=idx*2;                                                  
	c[idc]=br;c[idc+1]=bi;
}

extern "C" const char * CUDAset_carr(float br, float bi, float * c, int N)
{                                                                       
    hipError_t myerr;                                                \
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);          
	set_carr<<<nBlocks,blockSize>>>(br,bi,c,N);                        
  myerr=hipGetLastError();                                             
  if (myerr != hipSuccess)                                             
      return hipGetErrorString(myerr);                                 
  return 0;                                                             
}                                                                       


extern "C" unsigned long CUDAmaxSize() {
    int dev=0;
    hipGetDevice(&dev);
    struct hipDeviceProp_t prop;
    int status=hipGetDeviceProperties(&prop,dev);

    // return prop.maxThreadsPerBlock;  // 512
    // return prop.multiProcessorCount;   // 30
    // return prop.warpSize;   // 32
    // return prop.maxThreadsDim[0];   // 512  = max blocksize
    // return prop.maxGridSize[0];   // 65535  = max GridSize = max nBlocks?
    return ((long)prop.maxGridSize[0])*((long)prop.maxThreadsDim[0]);   // 65535  = max GridSize = max nBlocks?
}


__global__ void
arr_times_const_checkerboard(float*a,float b, float * c, int N, int sx,int sy,int sz)
{
	int ids=blockIdx.x*blockDim.x+threadIdx.x;   // which source array element do I have to deal with?
	if(ids>=N) return;  // not in range ... quit

	int px=(ids/2)%sx;   // my x pos
	int py=(ids/2)/sx;   // my y pos
    float minus1=(1-2*((px+py)%2));
	c[ids]=a[ids]*b*minus1;
}

extern "C" int CUDAarr_times_const_checkerboard(float * a, float b, float * c, int * sizes, int dims)  // multiplies with a constand and scrambles the array
{
    int sx=sizes[0],sy=1,sz=1;
    if (dims>1)
        sy=sizes[1];
    if (dims>2)
        sz=sizes[2];
    int N=sx*sy*sz*2;  // every pair will be processed exactly once
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);	// add extra block if N can't be divided by blockSize
	arr_times_const_checkerboard<<<nBlocks,blockSize>>>(a,b,c,N,sx,sy,sz);
	return 0;
}


/// cyclicly rotates datastack cyclic into positive direction in all coordinates by (dx,dy,dz) voxels
/// simple version with all processors dealing with exactly one element
__global__ void
rotate2(float*a,float b, float * c, int sx,int sy,int sz, int dx, int dy, int dz)
{
  int ids=(blockIdx.x*blockDim.x+threadIdx.x); // id of this processor
  int x=(ids + dx)%sx;  // advance by the offset steps along the chain
  int y=(ids/sx + dy)%sy;
  int z=(ids/(sx*sy) + dz)%sz;
  int idd=x+sx*y+sx*sy*z;
  if(ids>=sx*sy*sz) return;
  // float tmp = a[ids];
  // __syncthreads();             // nice try but does not work !
  c[idd] = b*a[ids];
}

/// cyclicly rotates datastack cyclic into positive direction in all coordinates by (dx,dy,dz) voxels
__global__ void
rotate(float*a,float b, float * c, int sx,int sy,int sz, int dx, int dy, int dz, int ux, int uy, int uz)
{
  int id=(blockIdx.x*blockDim.x+threadIdx.x); // id of this processor

  int Processes=blockDim.x * gridDim.x;
  int chains=ux*uy*uz; // total number of independent chains
  int N=sx*sy*sz;  // total size of array, has to be chains*length_of_chain
  int length=N/chains;  // chain length
  int steps=N/Processes;  // this is how many steps each processor has to do

  int step,nl,nx,ny,nz,x,y,z,i,idd;
  float swp, nswp;

//if (id != 0)   return;
//for (id=0;id<Processes;id++)
{
  step=steps*id;   // my starting step as the id times the number of steps
  nl=step%length;  // current position in chain length
  nx=(step/length)%ux;  // current position in unit cell x
  ny=(step/(length*ux))%uy;  // current position in unit cell y
  nz=(step/(length*ux*uy))%uz;  // current position in unit cell z
  i=0;

  //if (step/steps != 4 && step/steps != 5) return;

  while(nz<uz)
   {
      while(ny<uy)
        {
          while (nx<ux)
            {
            x=(nx+nl*dx)%sx;  // advance by the offset steps along the chain
            y=(ny+nl*dy)%sy;
            z=(nz+nl*dz)%sz;
            idd=x+sx*y+sx*sy*z;
            if (i < steps) {
                swp=a[idd]; 
                // a[idd]=a[idd]+0.1;
                __syncthreads();
            }
            while (nl<length-1)
                {
                  if (i > steps-1)
                    goto nextProcessor; // return;
                  if (step >= N)  // this thread has reached the end of the total data to process
                    goto nextProcessor; // return;
                  step++;
                  x = (x+dx)%sx; // new position
                  y = (y+dy)%sy;
                  z = (z+dz)%sz;
                  idd=x+sx*y+sx*sy*z;
                  if (i < steps-1) {
                    nswp=a[idd];
                    __syncthreads();
                    //a[idd]=a[idd]+0.1;
                    }

                  c[idd]=swp+0.1; // c[idd]+ny+0.1; // c[idd]+i; // swp+0.1; // c[idd]+(step/steps);
                  i++; // counts number of writes
                  if (i > steps-1)
                    goto nextProcessor; // return;
                  nl++;
                  if (i < steps) {
                  swp=nswp;
                  }
                }
            nx++; nl=0;
            //if (nx < ux) {
            x = (x+dx)%sx; // new position
            y = (y+dy)%sy;
            z = (z+dz)%sz;
            idd=x+sx*y+sx*sy*z;
            c[idd]=swp+0.1; // no need to save this value as this is the end of the line
            //}
            i++; 
            if (i > steps-1)
                goto nextProcessor; // return;
            // if (nx <ux) x=(x+1)%sx;
            }
        ny++;
        // if (ny <uy) y=(y+1)%sy;
        nx=0;x=0;
        }
    nz++;
    // if (nz <uz) z=(z+1)%sz;
    ny=0;y=0;
    }
nextProcessor:
nz=0;
}
return;
}

int gcd(int a, int b) // greatest commod divisor by recursion
{ 
   return ( b == 0 ? a : gcd(b, a % b) ); 
}

extern "C" int CUDAarr_times_const_rotate(float * a, float b, float * c, int * sizes, int dims, int complex,int direction)  // multiplies with a constand and cyclilcally rotates the array using the chain algorithm
{
    int sx=1,sy=1,sz=1;
    if (dims>0)
        sx=sizes[0];
    if (dims>1)
        {sx=sizes[0];sy=sizes[1];}
    if (dims>2)
        sz=sizes[2]; 

    int dx=(sx+direction*sx/2)%sx,dy=(sy+direction*sy/2)%sy,dz=(sz+direction*sz/2)%sz;  // how much to cyclically rotate
    if (complex) {sx=sx*2;dx=dx*2;}
    //printf("sx %d sy %d dx %d dy %d\n",sx,sy,dx,dy);

    // calculate the length of each swapping chain
    int ux=gcd(sx,dx);  // unit cell in x. Any repeat along y directions will be also a repeat in x. Chain length is sx/ux
    // int lx=sx/ux; // how many accesses to get one round in x
    int uy=gcd(((sx/ux)*dy%sy),sy); // how many times must the first chain be repeated to form a longer chain. This defines unit cell y
    int uz=gcd(((sy/uy)*dz%sz),sz); // similar for z
    int length=sx*sy*sz/(ux*uy*uz);  // chain length

    // in one dimension the gcd=greatest common divisor, would mean that one has to start task at position 0 ... gcd-1
    // in several dimensions even completing one round leaving a spacing at gcd does not mean that this is a complete loop
    // however it could be a complete loop. The number of steps that where performed in the lower dimension are s/gcd before reaching the beginning again
    // with the size of the dimension s. If we are at the same startingpoint in the next dimension the chain is complete.
    // So the number of times a super chain (in 2D) must be executed is sy/gcd(sy,s/gcd(sx,dx))
    int dev=0;
    hipGetDevice(&dev);
    struct hipDeviceProp_t prop;
    int status=hipGetDeviceProperties(&prop,dev);

    int m=1;
    if (ux>uy)
        m=ux;
    else
        m=uy;
    if (uz>m)
        m=uz;
    if (length>m)
        m=length;

    //int blockSize=1; // prop.warpSize; // ux*uy*uz;
    //int nBlocks=m;	// add extra block if N can't be divided by blockSize
    
    //    rotate<<<nBlocks,blockSize>>>(a,b,c,sx,sy,sz,dx,dy,dz,ux,uy,uz);  // get unit cell sizes

    int N=sx*sy*sz; // includes the space for coomplex numbers
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);	// add extra block if N can't be divided by blockSize
                                                                //    printf("BlockSize %d, ux %d, uy %d, uz %d\n",blockSize,ux,uy,uz);
    // unfortunately we have to do it out of place.
    if (a == c)
    {
        float * d =0;
        int status=hipMalloc((void **) &d, N*sizeof(float));
        hipMemcpy(d,a, N*sizeof(float),hipMemcpyDeviceToDevice);
        rotate2 <<<nBlocks,blockSize>>>(d,b,c,sx,sy,sz,dx,dy,dz);
        hipFree(d);
    }
    else
        rotate2 <<<nBlocks,blockSize>>>(a,b,c,sx,sy,sz,dx,dy,dz);  // get unit cell sizes

	return prop.maxThreadsPerBlock;
}



__global__ void
arr_times_const_scramble(float*a,float b, float * c, int sx,int sy,int sz, int ox, int oy, int oz)
{
	int pnum=blockIdx.x*blockDim.x+threadIdx.x;   // which source array element do I have to deal with?

	int px=pnum%(sx/2);   // my x pos of a complex number in the subarray
	int py=pnum/(sx/2);   // my y pos of a complex number
	if(px>=(sx/2) || py >= (sy/2)) return;  // not in range ... quit
    int ids=2*(px+py*sx);  /// offset to array start in floats
    int idd=2*((ox+px)+(oy+py)*sx);

    // echange two values using a tmp
    float tmpR = c[idd];
    float tmpI = c[idd+1];
    c[idd]=a[ids]; // (float)(ox+px); // 
    c[idd+1]=a[ids+1]; // (float)(oy+py); // 
    a[ids]=tmpR;
    a[ids+1]=tmpI;
}

__global__ void
array_copy(float*a, float * c, int mx, int my, int mz, int sx,int sy,int sz, int ox, int oy, int oz)  // copies between two memories with different strides
{
	int pnum=blockIdx.x*blockDim.x+threadIdx.x;   // which source array element do I have to deal with?

	int px=pnum%(sx/2);   // my x pos of a complex number in the subarray
	int py=pnum/(sx/2);   // my y pos of a complex number
	if(px>=sx || py >= (sy/2)) return;  // not in range ... quit
    int ids=2*(px+py*sx);  /// offset to array start in floats
    int idd=2*((ox+px)+(oy+py)*sx);

    // echange two values using a tmp
    float tmpR = c[idd];
    float tmpI = c[idd+1];
    c[idd]=a[ids]; // (float)(ox+px); // 
    c[idd+1]=a[ids+1]; // (float)(oy+py); // 
    a[ids]=tmpR;
    a[ids+1]=tmpI;
}


extern "C" int CUDAarr_times_const_scramble(float * a, float b, float * c, int * sizes, int dims)  // multiplies with a constand and scrambles the array
{
    int sx=sizes[0],sy=1,sz=1, iseven=1;
    if (sx%2 == 1) iseven=0;
    if (dims>1) {
        sy=sizes[1];
        if (sy%2 == 1) iseven=0;
        }

    if (dims>2) {
        sz=sizes[2];
        if (sz%2 == 1) iseven=0;
        }
    int N=sx*sy*sz*2;  // every pair will be processed exactly once
	int blockSize=BLOCKSIZE; int nBlocks=NBLOCKS(N,blockSize);	// add extra block if N can't be divided by blockSize

    if (! iseven)
        {
            float * tmp=0;
            hipMalloc((void **) &tmp,sizeof(tmp[0])*(1+sx/2)*(1+sy/2));
        }
	arr_times_const_scramble<<<nBlocks,blockSize>>>(a,b,c,sx,sy,sz,sx/2,sy/2,0);
	arr_times_const_scramble<<<nBlocks,blockSize>>>(a+2*(sx/2),b,c+2*(sx/2),sx,sy,sz,-sx/2,sy/2,0);
	return 0;
}


